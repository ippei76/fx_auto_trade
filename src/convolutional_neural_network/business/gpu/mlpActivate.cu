#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>

extern __global__ void kernelMlpActivate(const int mlpLayer);
extern __device__ float relu(const int outputIdx, const int miniBatchIdx, const int mlpLayer);
extern __global__ void kernelMlpActivateLastLayer(const int mlpLayer);
extern __device__ float softmax(const int outputIdx, const int outputIdxNums, const int miniBatchIdx, const int mlpLayer);

void mlpActivate(const int mlpLayer){

//	puts("mlpActivate start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	if(mlpLayer != getMlpOutputNumsNums() - 1){
		//カーネルの次元設定
		dim3 grid(getMlpOutputNums(mlpLayer), getMiniBatchNums()); //miniBatch毎の1ニューロンの出力を1ブロックとする。
		dim3 block(1, 1, 1); // 1ブロック1スレッド

		//次元チェック
		checkGridSize(grid);
		checkThreadSize(block);

//		gettimeofday(&t2, NULL);
		hipDeviceSynchronize();
		kernelMlpActivate<<<grid, block>>>(mlpLayer);
	}
	else{
		//カーネルの次元設定
		dim3 grid(getMlpOutputNums(mlpLayer), getMiniBatchNums()); //miniBatch毎の1ニューロンの出力を1ブロックとする。
		dim3 block(1, 1, 1); // 1ブロック1スレッド

		//次元チェック
		checkGridSize(grid);
		checkThreadSize(block);

//		gettimeofday(&t2, NULL);
		hipDeviceSynchronize();
		kernelMlpActivateLastLayer<<<grid, block>>>(mlpLayer);
	}
//	puts("kernelMlpActivate end.");
//	gettimeofday(&t3, NULL);

//	puts("mlpActivate end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelMlpActivate(const int mlpLayer){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;

	dMlpA[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)] = relu(outputIdx, miniBatchIdx, mlpLayer);
	/*
	if(outputIdx == 2){
		printf("mlpA(%d,%d,%d) = %f\n", outputIdx, miniBatchIdx, mlpLayer, dMlpA[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)]);
	}
	*/
}

__device__ float
relu(const int outputIdx, const int miniBatchIdx, const int mlpLayer){
	float val;
	val = getDMlpBn(outputIdx, miniBatchIdx, mlpLayer) *\
	      (0 < getDMlpBn(outputIdx, miniBatchIdx, mlpLayer));
	return(val);
}

__global__ void
kernelMlpActivateLastLayer(const int mlpLayer){
	int outputIdx = blockIdx.x;
	int outputIdxNums = gridDim.x;
	int miniBatchIdx = blockIdx.y;

	float val = softmax(outputIdx, outputIdxNums, miniBatchIdx, mlpLayer);
	dMlpA[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)] = val;
	//resultにも格納する
	dResult[getDim2Idx(outputIdx, miniBatchIdx, outputIdxNums)] = val;
	/*
	if(miniBatchIdx == 0){
		printf("soft_dResult(%d,%d,%d)[%f] = %f\n",outputIdx, miniBatchIdx, mlpLayer, getDMlpBn(outputIdx, miniBatchIdx, mlpLayer), dResult[getDim2Idx(outputIdx, miniBatchIdx, outputIdxNums)]);
	}
	*/
}

__device__ float
softmax(const int outputIdx, const int outputIdxNums, const int miniBatchIdx, const int mlpLayer){
	float val;
	float sumVal = 0;
	int i;
	//オーバーフローを防ぐために、最大値を求める。
	float maxMlpBn = getDMlpBn(0, miniBatchIdx, mlpLayer);
	for(i = 1; i < outputIdxNums; i++){
		float targetMlpBn = getDMlpBn(i, miniBatchIdx, mlpLayer);
		maxMlpBn = maxMlpBn * (maxMlpBn >= targetMlpBn) + targetMlpBn * (maxMlpBn < targetMlpBn);
	}

	//softmaxの分母を計算する。
	for(i = 0; i < outputIdxNums; i++){
		sumVal += expf(floatSubtraction(getDMlpBn(i, miniBatchIdx, mlpLayer), maxMlpBn, mlpLayer, __func__));
	}

	val = expf(floatSubtraction(getDMlpBn(outputIdx, miniBatchIdx, mlpLayer), maxMlpBn, mlpLayer, __func__)) / sumVal;
	/*
	if(miniBatchIdx == 0){
		printf("bn(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer,getDMlpBn(outputIdx, miniBatchIdx, mlpLayer));
		printf("sumval(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer,sumVal);
		printf("val(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer,val);
	}
	*/
	return(val);
}
