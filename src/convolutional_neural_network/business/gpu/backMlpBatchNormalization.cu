#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>
#include <commonFunc.cuh>

extern __global__ void kernelBackMlpBatchNormalization(const int mlpLayer);
extern __device__ float reluDelA_bn(const int outputIdx, const int miniBatchIdx, const int mlpLayer);

void backMlpBatchNormalization(const int mlpLayer){

//	puts("backMlpBatchNormalization start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//カーネルの次元設定
	dim3 grid(getMlpOutputNums(mlpLayer), getMiniBatchNums()); //miniBatch毎の1ニューロンの出力を1ブロックとする。
	dim3 block(1, 1, 1); // 1ブロック1スレッド

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

//	gettimeofday(&t2, NULL);
//	puts("kernelBackMlpBatchNormalization start.");
	hipDeviceSynchronize();
	kernelBackMlpBatchNormalization<<<grid, block>>>(mlpLayer);
//	puts("kernelBackMlpBatchNormalization end.");
//	gettimeofday(&t3, NULL);

//	puts("backMlpBatchNormalization end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelBackMlpBatchNormalization(const int mlpLayer){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;

	dMlpBnBack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)]\
		= getDMlpABack(outputIdx, miniBatchIdx, mlpLayer) * reluDelA_bn(outputIdx, miniBatchIdx, mlpLayer);
	/*
	if(miniBatchIdx == 2){
		printf("mlpABack(%d,%d,%d):%f\n", outputIdx, miniBatchIdx, mlpLayer, getDMlpABack(outputIdx, miniBatchIdx, mlpLayer));
		printf("mlpBn(%d,%d,%d):%f\n", outputIdx, miniBatchIdx, mlpLayer, getDMlpBn(outputIdx, miniBatchIdx, mlpLayer));
		printf("mlpBnBack(%d,%d,%d):%f\n", outputIdx, miniBatchIdx, mlpLayer, dMlpBnBack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)]);
	}
	*/
}

__device__ float
reluDelA_bn(const int outputIdx, const int miniBatchIdx, const int mlpLayer){
	float val;
	val = (0 < getDMlpBn(outputIdx, miniBatchIdx, mlpLayer));
	return(val);
}
