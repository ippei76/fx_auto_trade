#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>
#include <commonFunc.cuh>

__global__ void kernelBackCnnActivateProp(const int cnnLayer);

void backCnnActivate(const int cnnLayer){

//	puts("backCnnActivate start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//カーネルの次元設定
	dim3 gridProp(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 blockProp(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1); // 1ブロックp_x * p_yスレッド

	//次元チェック
	checkGridSize(gridProp);
	checkThreadSize(blockProp);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelBackCnnActivateProp start.");
	hipDeviceSynchronize();
	kernelBackCnnActivateProp<<<gridProp, blockProp>>>(cnnLayer);
//	puts("kernelBackCnnActivateProp end.");
//	gettimeofday(&t3, NULL);

//	puts("backCnnActivate end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelBackCnnActivateProp(const int cnnLayer){
	int cnnWba_x = threadIdx.x;
	int cnnWba_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int cnnP_x = cnnWba_x / getCCnnPooling_xNums();
	int cnnP_y = cnnWba_y / getCCnnPooling_yNums();

	float tgtCnnA = getDCnnA(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer);
	float tgtCnnP = getDCnnP(cnnP_x, cnnP_y, outputIdx, miniBatchIdx, cnnLayer);

	//更新
	dCnnABack[getDCnnWbaIdx(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer)] = getDCnnPBack(cnnP_x, cnnP_y, outputIdx, miniBatchIdx, cnnLayer) * (tgtCnnA == tgtCnnP);
	/*
	if(outputIdx == 1 && miniBatchIdx == 1){
		printf("cnnABack(%d,%d,%d,%d,%d):%f\n",cnnWba_x,cnnWba_y,outputIdx,miniBatchIdx,cnnLayer,dCnnABack[getDCnnWbaIdx(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer)]);
	}
	*/
}
