#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>
#include <commonFunc.cuh>
#include <sys/time.h>

//extern void dynamicAllocateDeviceMemory(float *cnnW, const int cnnWDataNums, float *mlpW, const int mlpWDataNums);
extern void cnnConvolution(const int cnnLayer);
extern void cnnBatchNormalization(const int cnnLayer);
extern void cnnActivate(const int cnnLayer);
extern void cnnPooling(const int cnnLayer);
extern void mlpMultiply(const int mlpLayer);
extern void mlpBatchNormalization(const int mlpLayer);
extern void mlpActivate(const int mlpLayer);
extern void lossFunc(const int mlpOutputNumsLastIdx);
extern void backMlpActivate(const int mlpLayer);
extern void backMlpBatchNormalization(const int mlpLayer);
extern void backMlpMultiply(const int mlpLayer);
extern void backCnnPMlpW(const int cnnLayer);
extern void backCnnPooling(const int cnnLayer);
extern void backCnnActivate(const int cnnLayer);
extern void backCnnBatchNormalization(const int cnnLayer);
extern void backCnnConvolution(const int cnnLayer);
extern void backCnnSv(const int cnnLayer);
extern void inferenceBnMeanVar2(const int episode, float *infCnnBnMean, float *infCnnBnVar2, float *infMlpBnMean, float *infMlpBnVar2);
extern void cnnBatchNormalization_culcurationAveVar2(const int cnnLayer);

void cnnForwardPropagation(){

//	struct timeval t1, t2, t3, t4, t5, t6, t7;
//	gettimeofday(&t1, NULL);
//	printTime(t1,t2,t3);

	int cnnLayer;
	for(cnnLayer = 0; cnnLayer < getCnnOutputNumsNums(); cnnLayer ++){
//		printf("\n\ncnnLayer:%d\n\n",cnnLayer);
		//inputDataに関するチェック
		//checkInputDataSize(inputDataSize);
		//checkInput_xyNums(input_xNums, input_yNums);
		//checkInputChannelNums(inputChannelNums);

		cnnConvolution(cnnLayer);
		cnnBatchNormalization(cnnLayer);
		cnnActivate(cnnLayer);
		cnnPooling(cnnLayer);
	}
}

void mlpForwardPropagation(){

	int mlpLayer;
	for(mlpLayer = 0; mlpLayer < getMlpOutputNumsNums(); mlpLayer++){
//		printf("\n\nmlpLayer:%d\n\n",mlpLayer);

		mlpMultiply(mlpLayer);
		mlpBatchNormalization(mlpLayer);
		mlpActivate(mlpLayer);
	}
}

void mlpBackPropagation(){

	//mlp最終層取得
	int mlpOutputNumsLastIdx = getMlpOutputNumsNums() - 1;
	int mlpLayer;

	for(mlpLayer = mlpOutputNumsLastIdx; mlpLayer >= 0; mlpLayer--){
//		printf("\n\nmlpLayerBack:%d\n\n",mlpLayer);
		if(mlpLayer == mlpOutputNumsLastIdx){
			//mlp最終層のmlpBnを求める。
			lossFunc(mlpOutputNumsLastIdx);
		}
		else{
			backMlpActivate(mlpLayer);
			backMlpBatchNormalization(mlpLayer);
		}
		backMlpMultiply(mlpLayer);
	}

}

void cnnBackPropagation(){

	//cnn最終層取得
	int cnnOutputNumsLastIdx = getCnnOutputNumsNums() - 1;
	int cnnLayer;

	for(cnnLayer = cnnOutputNumsLastIdx; cnnLayer >= -1; cnnLayer--){
//		printf("\n\ncnnLayerBack:%d\n\n",cnnLayer);
		if(cnnLayer == cnnOutputNumsLastIdx){
			//cnn最終層のcnnPとmlp0層のmlpWを更新する。
			backCnnPMlpW(cnnLayer); //cnnLayerを渡していることに注意
		}
		else{
			if(cnnLayer != -1){
				backCnnPooling(cnnLayer);
			}
			else{
				//最後に0層のwを更新する。
				backCnnSv(cnnLayer);
				break;
			}
		}
		backCnnActivate(cnnLayer);
		backCnnBatchNormalization(cnnLayer);
		backCnnConvolution(cnnLayer);

	}
}

void getDeviceResult(){

	//GPUメモリより、最終アウトプットをメモリにコピー
	int resultDataSize = getMlpOutputNums(getMlpOutputNumsNums() - 1) * getMiniBatchNums() * sizeof(float);
	gpuErrchk(hipMemcpyFromSymbol(result, HIP_SYMBOL(dResult), resultDataSize));
	/*
	int i;
	for(i = 0; i< resultDataSize / sizeof(float);i++){
		printf("result[%d]=%f\n",i,result[i]);
	}
	*/

}

void getScore(){
	int outputIdx, miniBatchIdx;
	E = 0.0;
	for(miniBatchIdx = 0; miniBatchIdx < getMiniBatchNums(); miniBatchIdx++){
		for(outputIdx = 0; outputIdx < getMlpOutputNums(getMlpOutputNumsNums() - 1); outputIdx++){
			//softmaxでは、分母に無限大が発生する可能性がある。それをここで検知する。
		//	printf("koko%f ",getResult(outputIdx, miniBatchIdx));
			checkInfNan(getResult(outputIdx, miniBatchIdx), "resultValues");
			if(getResult(outputIdx, miniBatchIdx) > 0){
				E += (-1) * getTeachOut(outputIdx, miniBatchIdx) * logf(getResult(outputIdx, miniBatchIdx));
			}
			else{
				E += (-1) * getTeachOut(outputIdx, miniBatchIdx) * logf(getResult(outputIdx, miniBatchIdx) + getBnEps());
			}
			checkInfNan(E, "EValues");
		//	printf("LOG:result[%d,%d]=%f, teachOut[%d,%d]=%f\n",outputIdx,miniBatchIdx,getResult(outputIdx, miniBatchIdx), outputIdx,miniBatchIdx,getTeachOut(outputIdx, miniBatchIdx));
		}
	}
	E = E / getMiniBatchNums();
	printf("LOG:E=%f\n",E);
}

bool breakCheck(){
	if(E < 0.003){
		printf("LOG: E is very small. break.\n");
		return(true);
	}
	return(false);
}

void attenuationLearningRate(const int episode){
	if((episode + 1) % 500 == 0){
		float newLearningRate = getLearningRate() * 0.99;
		setLearningRate(newLearningRate);
		gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cLearningRate), &learningRate, sizeof(float)));
	}
}

void jnaExecuteOnline(){

	cnnForwardPropagation();
	mlpForwardPropagation();
	getDeviceResult();

}
void jnaExecuteTraining(){

//	struct timeval t1, t2, t3, t4, t5, t6, t7;
//	gettimeofday(&t1, NULL);
	cnnForwardPropagation();
//	gettimeofday(&t2, NULL);
	mlpForwardPropagation();
//	gettimeofday(&t3, NULL);
	getDeviceResult();
//	gettimeofday(&t4, NULL);
	getScore();
//	gettimeofday(&t5, NULL);
	mlpBackPropagation();
//	gettimeofday(&t6, NULL);
	cnnBackPropagation();
//	gettimeofday(&t7, NULL);

//	printTime(t1,t2,t3);
//	printTime(t3,t4,t5);
//	printTime(t5,t6,t7);
//	exit(2);
}

int getAllDataIdx(const int allDataNums, const int oneTeachOutDataNumsNoMiniBatch, const float *teachOutAll){
	int targetIdx = -1;
	targetIdx = getRandomInt(0, allDataNums);
	while(true){
		/*
		printf("targetIdx:%d{",targetIdx);
		int i;
		for(i = 0; i < oneTeachOutDataNumsNoMiniBatch; i++){
			printf("%f ",teachOutAll[targetIdx * oneTeachOutDataNumsNoMiniBatch + i]);
		}
		*/
		//wait(=0,0,1)がteachOutの場合は除外する。
		if(teachOutAll[targetIdx * oneTeachOutDataNumsNoMiniBatch + 2] == 1){
		//if(teachOutAll[targetIdx * oneTeachOutDataNumsNoMiniBatch + 0] == 1){
			//break;
			targetIdx = getRandomInt(0, allDataNums);

		}
		else{
			//printf("skip(%d)\n",targetIdx);
			break;
		}
	}
	return(targetIdx);
}

void jnaExecuteBeforeTraining(const float *svAll, const float *teachOutAll, const int oneSvDataNums, const int oneTeachOutDataNums, const int allDataNums){
	//教師データの中から、miniBatchNums数選び、dSv,teachOutにセットする。
	//oneSV,oneTeachOutはminiBatchを含んでいる。
	int svIdx = 0;
	int teachOutIdx = 0;
	int miniBatchIdx, allDataIdx, i, j;
	int oneSvDataSize = oneSvDataNums * sizeof(float);
	int oneTeachOutDataSize = oneTeachOutDataNums * sizeof(float);
	int oneSvDataNumsNoMiniBatch = oneSvDataNums / getMiniBatchNums();
	int oneTeachOutDataNumsNoMiniBatch = oneTeachOutDataNums / getMiniBatchNums();
	float *sv; //teachOutはグローバル変数を使う。
	//sv,teachOutの動的確保
	sv = (float *)malloc(oneSvDataSize);
	teachOut = (float *)malloc(oneTeachOutDataSize);
	for(miniBatchIdx = 0; miniBatchIdx < getMiniBatchNums(); miniBatchIdx++){
		//ランダム選択
		allDataIdx = getAllDataIdx(allDataNums, oneTeachOutDataNumsNoMiniBatch, teachOutAll);
		//printf("%d ",allDataIdx);
		//printf("LOG:allDataIdx=%d\n",allDataIdx);
		for(i = 0; i < oneSvDataNumsNoMiniBatch; i++){	//miniBatchを含まない1データ
			sv[svIdx] = svAll[allDataIdx * oneSvDataNumsNoMiniBatch + i];
			//sv[svIdx] = svIdx;
			//sv[svIdx] = 1;
//			printf("sv[%d]=%f\n",svIdx,sv[svIdx]);
			svIdx++;
		}
		for(j = 0; j < oneTeachOutDataNumsNoMiniBatch; j++){	//miniBatchを含まない1データ
			teachOut[teachOutIdx] = teachOutAll[allDataIdx * oneTeachOutDataNumsNoMiniBatch + j];
			/*
			if(j%oneTeachOutDataNumsNoMiniBatch==0){
				teachOut[teachOutIdx] = 1;
			}
			else{
				teachOut[teachOutIdx] = 0;
			}
			*/
	//		printf("teachOutAll[%d]=%f\n",allDataIdx * oneTeachOutDataNumsNoMiniBatch + j,teachOut[teachOutIdx]);
			teachOutIdx++;
		}
	}
	/*
	for(i=0; i < oneTeachOutDataNumsNoMiniBatch * getMiniBatchNums(); i++){
		printf("%d = %f , ",i,teachOut[i]);
	}
	*/
	//GPUチェック。
	checkSvTeachOutDataNums(oneSvDataNums, oneTeachOutDataNums);
	//GPUへコピー
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dSv), sv, oneSvDataSize));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dTeachOut), teachOut, oneTeachOutDataSize));
}

void jnaExecuteBeforeOnline(const float *sv_arg, const int oneSvDataNums, const float *cnnBnAveMean_arg, const float *cnnBnAveVar2_arg, const float *mlpBnAveMean_arg, const float *mlpBnAveVar2_arg){

	int oneSvDataSize = oneSvDataNums * sizeof(float);
	//svをGPUへコピー
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dSv), sv_arg, oneSvDataSize));
	//トレーニングにて算出した平均分散をコンスタントメモリにコピーする。
	int cnnBnMeanVar2DataSize = sizeof(float) * getCnnOutputNumsSum();
	int mlpBnMeanVar2DataSize = sizeof(float) * getMlpOutputNumsSum();
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnBnAveMean), cnnBnAveMean_arg, cnnBnMeanVar2DataSize));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnBnAveVar2), cnnBnAveVar2_arg, cnnBnMeanVar2DataSize));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cMlpBnAveMean), mlpBnAveMean_arg, mlpBnMeanVar2DataSize));
	gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cMlpBnAveVar2), mlpBnAveVar2_arg, mlpBnMeanVar2DataSize));

}

void jnaExecuteAfterTraining(float *cnnW, float *mlpW, float *cnnBnGamma, float *cnnBnBeta, float *mlpBnGamma, float *mlpBnBeta){

	int cnnWDataSize = getCnnWDataNums() * sizeof(float);
	int mlpWDataSize = getMlpWDataNums() * sizeof(float);
	int cnnBnGammaBetaSize = getCnnOutputNumsSum() * sizeof(float);
	int mlpBnGammaBetaSize = getMlpOutputNumsSum() * sizeof(float);

	//学習済みのcnnW,mlpW,cnnBnGamma,cnnBnBeta,mlpBnGamma,mlpBnBetaをGPUから移送する。
	gpuErrchk(hipMemcpyFromSymbol(cnnW, HIP_SYMBOL(dCnnW), cnnWDataSize));
	gpuErrchk(hipMemcpyFromSymbol(mlpW, HIP_SYMBOL(dMlpW), mlpWDataSize));
	gpuErrchk(hipMemcpyFromSymbol(cnnBnGamma, HIP_SYMBOL(dCnnBnGamma), cnnBnGammaBetaSize));
	gpuErrchk(hipMemcpyFromSymbol(cnnBnBeta, HIP_SYMBOL(dCnnBnBeta), cnnBnGammaBetaSize));
	gpuErrchk(hipMemcpyFromSymbol(mlpBnGamma, HIP_SYMBOL(dMlpBnGamma), mlpBnGammaBetaSize));
	gpuErrchk(hipMemcpyFromSymbol(mlpBnBeta, HIP_SYMBOL(dMlpBnBeta), mlpBnGammaBetaSize));

	/*終了処理*/
	hipDeviceReset();

}
void jnaExecuteAfterOnline(){

	/*終了処理*/
	hipDeviceReset();

}

void cnnMain(
		int sv_xNums_arg, int sv_yNums_arg, int miniBatchNums_arg, int svChannelNums_arg, float *sv_arg, float *svAll, float *teachOut_arg, float *teachOutAll,

		int *cnnOutputNums_arg, int cnnOutputNumsNums_arg, 

		float *cnnBnBeta_arg, float *cnnBnGamma_arg, float bnEps_arg,

		int cnnW_xNums_arg, int cnnW_yNums_arg, float *cnnW_arg, 

		int cnnPooling_xNums_arg, int cnnPooling_yNums_arg, 

		int *cnnWba_xNums_arg, int *cnnWba_yNums_arg, 

		int *cnnP_xNums_arg, int *cnnP_yNums_arg,

		int *mlpOutputNums_arg, int mlpOutputNumsNums_arg,

		float *mlpBnBeta_arg, float *mlpBnGamma_arg,

		float *mlpW_arg,

		int stepNums, int episodeNums,

		int oneSvDataNums, int oneTeachOutDataNums, int allDataNums,

		int cnnWbaDataNums, int cnnPDataNums, int mlpWbaDataNums,

		int cnnWDataNums_arg, int cnnBnBetaGammaDataNums,

		int mlpWDataNums_arg, int mlpBnBetaGammaDataNums,

		float *infCnnBnMean, float *infCnnBnVar2, float *infMlpBnMean, float *infMlpBnVar2,

		//トレーニング時には、使用しない。
		float *cnnBnAveMean, float *cnnBnAveVar2, float *mlpBnAveMean, float *mlpBnAveVar2,

		//「トレーニング:0」「Online:1」
		int execFlg_arg, float *result_arg, float learningRate_arg

		){
			puts("cnnMain start.");

			//CPU側のデータ確保
			//output
			setCnnOutputNums(cnnOutputNums_arg);
			setCnnOutputNumsNums(cnnOutputNumsNums_arg);
			//sv,teachOut
			setSv_xNums(sv_xNums_arg);
			setSv_yNums(sv_yNums_arg);
			setSvChannelNums(svChannelNums_arg);
			//setSv(sv_arg);
			setTeachOut(teachOut_arg);

			//cnn
			setCnnBnBeta(cnnBnBeta_arg);
			setCnnBnGamma(cnnBnGamma_arg);
			setBnEps(bnEps_arg);

			setCnnW_xNums(cnnW_xNums_arg);
			setCnnW_yNums(cnnW_yNums_arg);
			setCnnWDataNums(cnnWDataNums_arg);
			setCnnW(cnnW_arg); //kokoiru???

			setCnnPooling_xNums(cnnPooling_xNums_arg);
			setCnnPooling_yNums(cnnPooling_yNums_arg);

			setCnnWba_xNums(cnnWba_xNums_arg);
			setCnnWba_yNums(cnnWba_yNums_arg);

			setCnnP_xNums(cnnP_xNums_arg);
			setCnnP_yNums(cnnP_yNums_arg);

			//mlp
			setMlpOutputNums(mlpOutputNums_arg);
			setMlpOutputNumsNums(mlpOutputNumsNums_arg);

			setMlpWDataNums(mlpWDataNums_arg);

			setMlpBnBeta(mlpBnBeta_arg);
			setMlpBnGamma(mlpBnGamma_arg);

			setMiniBatchNums(miniBatchNums_arg);

			setExecFlg(execFlg_arg);

			//result
			setResult(result_arg);

			//other
			setLearningRate(learningRate_arg);

			//GPUメモリのチェック(ノード変数)
			//sv,teachOutはBeforeで実施
			//cnn,mlp,result
			checkNodeValues(cnnWbaDataNums, cnnPDataNums, mlpWbaDataNums, getMlpOutputNums(getMlpOutputNumsNums() - 1));

			//GPUメモリのチェック(学習変数)
			//cnn,mlp
			checkLearnValues(cnnWDataNums, cnnBnBetaGammaDataNums, mlpWDataNums, mlpBnBetaGammaDataNums);
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dCnnW), cnnW_arg, cnnWDataNums * sizeof(float)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dCnnBnBeta), cnnBnBeta_arg, cnnBnBetaGammaDataNums * sizeof(float)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dCnnBnGamma), cnnBnGamma_arg, cnnBnBetaGammaDataNums * sizeof(float)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dMlpW), mlpW_arg, mlpWDataNums * sizeof(float)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dMlpBnBeta), mlpBnBeta_arg, mlpBnBetaGammaDataNums * sizeof(float)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(dMlpBnGamma), mlpBnGamma_arg, mlpBnBetaGammaDataNums * sizeof(float)));

			//各種値の代入
			//sv
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cSvChannelNums), &svChannelNums_arg, sizeof(int)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cSv_xNums), &sv_xNums_arg, sizeof(int)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cSv_yNums), &sv_yNums_arg, sizeof(int)));
			//ノード数
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnOutputNums), cnnOutputNums_arg, sizeof(int) * getCnnOutputNumsNums()));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cMlpOutputNums), mlpOutputNums_arg, sizeof(int) * getCnnOutputNumsNums()));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnOutputNumsNums), &cnnOutputNumsNums_arg, sizeof(int)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cMlpOutputNumsNums), &mlpOutputNumsNums_arg, sizeof(int)));
			//出力(wba,p)
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnWba_xNums), cnnWba_xNums, sizeof(int) * getCnnOutputNumsNums()));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnWba_yNums), cnnWba_yNums, sizeof(int) * getCnnOutputNumsNums()));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnP_xNums), cnnP_xNums, sizeof(int) * getCnnOutputNumsNums()));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnP_yNums), cnnP_yNums, sizeof(int) * getCnnOutputNumsNums()));
			//cnnW
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnW_xNums), &cnnW_xNums, sizeof(int)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnW_yNums), &cnnW_yNums, sizeof(int)));
			//cnnPooling
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnPooling_xNums), &cnnPooling_xNums, sizeof(int)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cCnnPooling_yNums), &cnnPooling_yNums, sizeof(int)));
			//その他
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cMiniBatchNums), &miniBatchNums_arg, sizeof(int)));
			gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(cLearningRate), &learningRate, sizeof(float)));

			//Constant Memory チェック
			checkConstantMemory();

			//各種制限値チェック
			checkMiniBatchNums(getMiniBatchNums());
			checkOutputNums(cnnOutputNums, getCnnOutputNumsNums(), mlpOutputNums, getMlpOutputNumsNums());
			checkW_xyNums(getCnnW_xNums(), getCnnW_yNums());

			//Online
			if(getExecFlg() == getExecFlgOnline()){
				puts("Online start.");
				jnaExecuteBeforeOnline(sv_arg, oneSvDataNums, cnnBnAveMean, cnnBnAveVar2, mlpBnAveMean, mlpBnAveVar2);
				jnaExecuteOnline();
				jnaExecuteAfterOnline();
			}
			//Trainig
			else if(getExecFlg() == getExecFlgTraining()){
				puts("Training start.");
				int episode, step;
				for(episode = 0; episode < episodeNums; episode++){
					//struct timeval t1, t2, t3, t4;
					//gettimeofday(&t1, NULL);
					jnaExecuteBeforeTraining(svAll, teachOutAll, oneSvDataNums, oneTeachOutDataNums, allDataNums);
					//gettimeofday(&t2, NULL);
					for(step = 0; step < stepNums; step++){
						printf("LOG:episode=%d, step=%d, learningRate=%f\n",episode,step,getLearningRate());
						jnaExecuteTraining();
						/*
						if(step == 5){
							exit(2);
						}
						*/
						if(breakCheck() == true){
							break;
						}
						/*
						if(step == 4){
							printCnnMlpW();
							printCnnMlpBnBeta();
							printCnnMlpBnGamma();
							exit(2);
						}
						*/
					}
					//gettimeofday(&t3, NULL);
					//batch normalizationの結果をinfBnMean,Var2にコピーする。
					inferenceBnMeanVar2(episode, infCnnBnMean, infCnnBnVar2, infMlpBnMean, infMlpBnVar2);
					attenuationLearningRate(episode);
					//printf("timeRec\n");
					//printTime(t1,t2,t3);
					//exit(2);
				}
				jnaExecuteAfterTraining(cnnW_arg, mlpW_arg, cnnBnGamma_arg, cnnBnBeta_arg, mlpBnGamma_arg, mlpBnBeta_arg);
			}
			else{
				printf("execFlg error. (execFlg=%d)\n", getExecFlg());
			}
		}
