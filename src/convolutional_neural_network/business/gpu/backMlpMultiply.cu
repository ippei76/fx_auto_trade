#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

extern __global__ void kernelBackMlpMultiplyProp(const int mlpLayer, const float bnEps);
extern __global__ void kernelBackMlpMultiplyUpdate(const int mlpLayer, const int miniBatchIdxNums, const float bnEps);

void backMlpMultiply(const int mlpLayer){

//	puts("backMlpMultiply start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//ブロック・スレッド定義
	dim3 gridProp(getMlpOutputNums(mlpLayer), 1);
	dim3 blockProp(getMiniBatchNums(), 1, 1);
	dim3 gridUpdate(getMlpOutputNums(mlpLayer), 1);
	dim3 blockUpdate(1, 1, 1);

	//次元チェック
	checkGridSize(gridProp);
	checkThreadSize(blockProp);
	checkGridSize(gridUpdate);
	checkThreadSize(blockUpdate);

	//シェアードメモリ確保
	int miniBatchDataSize = sizeof(float) * getMiniBatchNums();
	int sharedSizeMlpBnTmp = miniBatchDataSize; 

	//シェアードメモリチェック
	checkSharedMemorySize(sharedSizeMlpBnTmp);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelBackMlpMultiply start");
	hipDeviceSynchronize();
	kernelBackMlpMultiplyProp<<<gridProp, blockProp, sharedSizeMlpBnTmp>>>(mlpLayer, getBnEps());
//	puts("kernelBackMlpMultiply end");
//	puts("kernelBackMlpMultiplyUpdate start");
	hipDeviceSynchronize();
	kernelBackMlpMultiplyUpdate<<<gridUpdate, blockUpdate>>>(mlpLayer, getMiniBatchNums(), getBnEps());
//	puts("kernelBackMlpMultiplyUpdate end");
//	gettimeofday(&t3, NULL);

//	puts("backMlpMultiply end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelBackMlpMultiplyProp(const int mlpLayer, const float bnEps){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = threadIdx.x;
	int miniBatchIdxNums = blockDim.x;
	int z;
	float mean, var2;
	float del2Tmp = 0;
	float del3Tmp = 0;

	//平均値を取得
	mean = getDMlpBnMean(outputIdx, mlpLayer);
	//分散を取得
	var2 = getDMlpBnVar2(outputIdx, mlpLayer);
	//printf("mean[%d]:%f\n", outputIdx , mean);
	//printf("var2[%d]:%f\n", outputIdx , var2);

	//シェアードメモリにmlpBnTmpを代入する。
	extern __shared__ float sMlpBnTmp[];
	sMlpBnTmp[miniBatchIdx] = (getDMlpWb(outputIdx, miniBatchIdx, mlpLayer) - mean) * powf((var2 + bnEps), -0.5f);
	__syncthreads();

	//mlpWb更新要素の計算
	for(z = 0; z < miniBatchIdxNums; z++){
		//del2Tmp += getDMlpBnBack(outputIdx, z, mlpLayer) * sMlpBnTmp[getDim2Idx(outputIdx, z, outputIdxNums)];
		del2Tmp += getDMlpBnBack(outputIdx, z, mlpLayer) * sMlpBnTmp[z];
		/*
		if(outputIdx == 1){
			printf("mlpBnBack(%d,%d):%f\n", outputIdx, z, getDMlpBnBack(outputIdx, z, mlpLayer));
			printf("mlpWb(%d,%d):%f\n", outputIdx, z, getDMlpWb(outputIdx, z, mlpLayer));
			printf("sMLp(%d,%d):%f\n", outputIdx, z, sMlpBnTmp[z]);
			printf("mean[%d]:%f\n", outputIdx , mean);
			printf("var2[%d]:%f\n", outputIdx , var2);
		}
		*/
		del3Tmp += getDMlpBnBack(outputIdx, z, mlpLayer);
	}
	float del1 = miniBatchIdxNums * getDMlpBnBack(outputIdx, miniBatchIdx, mlpLayer);
	float del2 = del2Tmp * sMlpBnTmp[miniBatchIdx];
	float del3 = del3Tmp;
	//float subtractDel = floatSubtraction(floatSubtraction(del1, del2), del3);
	float subtractDel = del1 - del2 - del3;

	/*
	if(outputIdx == 1){
		printf("del1(%d,%d):%f\n", outputIdx, miniBatchIdx, del1);
		printf("del2(%d,%d):%f\n", outputIdx, miniBatchIdx, del2);
		printf("del3(%d,%d):%f\n", outputIdx, miniBatchIdx, del3);
	}
	*/

	dMlpWbBack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)] =\
		subtractDel * getDMlpBnGamma(outputIdx, mlpLayer) * powf((var2 + bnEps), -0.5f) / miniBatchIdxNums;
//	if(miniBatchIdx == 2){
//		printf("subtractDel(%d,%d):%f\n", outputIdx, miniBatchIdx, subtractDel);
	//	printf("mlpWbBack(%d,%d):%f\n", outputIdx, miniBatchIdx, dMlpWbBack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)]);
//	}
}

__global__ void
kernelBackMlpMultiplyUpdate(const int mlpLayer, const int miniBatchIdxNums, const float bnEps){

	int outputIdx = blockIdx.x;
	int z;
	float mean, var2, bnTmp;
	float sumGamma = 0, sumBeta = 0;

	//平均値を取得
	mean = getDMlpBnMean(outputIdx, mlpLayer);
	//分散を取得
	var2 = getDMlpBnVar2(outputIdx, mlpLayer);

	//更新値を計算
	for(z = 0; z < miniBatchIdxNums; z++){
		/*
		if(z == 0){
			printf("mlpBackMean[%d][%d]:%f\n", outputIdx , mlpLayer, mean);
			printf("mlpBackVar2[%d][%d]:%f\n", outputIdx , mlpLayer, var2);
		}
		*/
//		printf("mlpBnBack(%d,%d):%f  ", outputIdx, z, getDMlpBnBack(outputIdx, z, mlpLayer));
//		printf("mlpWb(%d,%d):%f  ", outputIdx, z, getDMlpWb(outputIdx, z, mlpLayer));
		bnTmp = (getDMlpWb(outputIdx, z, mlpLayer) - mean) / powf((var2 + bnEps), 0.5f);
		sumGamma += getDMlpBnBack(outputIdx, z, mlpLayer) * bnTmp;
		sumBeta += getDMlpBnBack(outputIdx, z, mlpLayer);
	}
	//更新
	dMlpBnGamma[getDMlpBnMeanVar2Idx(outputIdx, mlpLayer)] -= sumGamma * getCLearningRate();
	dMlpBnBeta[getDMlpBnMeanVar2Idx(outputIdx, mlpLayer)] -= sumBeta * getCLearningRate();
	//if(outputIdx == 1){
//		printf("mlpgamma(%d):%f  ", outputIdx, dMlpBnGamma[getDMlpBnMeanVar2Idx(outputIdx, mlpLayer)]);
//		printf("mlpbeta(%d):%f  ", outputIdx, dMlpBnBeta[getDMlpBnMeanVar2Idx(outputIdx, mlpLayer)]);
//	}
}
