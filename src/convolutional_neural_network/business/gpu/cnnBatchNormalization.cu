#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

extern __global__ void kernelCnnBatchNormalizationTraining(int cnnLayer, float bnEps);
extern __global__ void kernelCnnBatchNormalizationOnline(int cnnLayer, float bnEps);

void cnnBatchNormalization(const int cnnLayer){

//	puts("cnnBatchNormalization start.");
	//struct timeval t1, t2, t3;

	//ブロック・スレッド定義
	dim3 grid(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 block(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

	//カーネル起動
//	puts("kernelCnnBatchNormalization start");
	if(getExecFlg() == getExecFlgTraining()){
	//	gettimeofday(&t1, NULL);
	//	gettimeofday(&t2, NULL);
		cnnBatchNormalization_culcurationAveVar2(cnnLayer);
		hipDeviceSynchronize();
		kernelCnnBatchNormalizationTraining<<<grid, block>>>(cnnLayer, getBnEps());
	//	gettimeofday(&t3, NULL);
	}
	else{
//		puts("online batchNormalization");
		hipDeviceSynchronize();
		kernelCnnBatchNormalizationOnline<<<grid, block>>>(cnnLayer, getBnEps());
	}
//	puts("kernelCnnBatchNormalization end");

//	puts("cnnBatchNormalization end.");
//	printTime(t1,t2,t3);

}

__global__ void kernelCnnBatchNormalizationTraining(int cnnLayer, float bnEps){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	float mean, var2, bnTmp;

	//	printf("aaOK");
	mean = dCnnBnMean[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)];
	var2 = dCnnBnVar2[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)];
	/*
	if(miniBatchIdx == 0 && wba_x == 0 && wba_y == 0){
//		printf("mean:%f",mean);
		printf("var2:%f",var2);
	}
	*/
	/*
	//平均を算出：miniBatch * wba_yNums * wba_xNums
	int x,y,z;
	float testmean,testvar2,tmp;
	float sumMean = 0;
	float sumVar2 = 0;
	int wba_xNums = blockDim.x;
	int wba_yNums = blockDim.y;
	int miniBatchIdxNums = gridDim.y;
	for(z = 0; z < miniBatchIdxNums; z++){
		for(y = 0; y < wba_yNums; y++){
			for(x = 0; x < wba_xNums; x++){
				tmp = getDCnnWb(x, y, outputIdx, z, cnnLayer);
				sumMean = tmp + sumMean;
			}
		}
	}
	testmean = sumMean / (miniBatchIdxNums * wba_yNums * wba_xNums);

	//分散を算出
	for(z = 0; z < miniBatchIdxNums; z++){
		for(y = 0; y < wba_yNums; y++){
			for(x = 0; x < wba_xNums; x++){
				tmp = getDCnnWb(x, y, outputIdx, z, cnnLayer);
				sumVar2 = powf(floatSubtraction(tmp, mean), 2) + sumVar2;
			}
		}
	}
	testvar2 = sumVar2 / (miniBatchIdxNums * wba_yNums * wba_xNums);
	printf("%d:mean:%f(%f)\n",cnnLayer,mean,testmean);
	printf("%d:var2:%f(%f)\n",cnnLayer,var2,testvar2);
	*/

	bnTmp = (getDCnnWb(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer) - mean) / powf((var2 + bnEps), 0.5f);
	if(powf((var2 + bnEps), 0.5f) <=0 ){
		printf("cnnkoko:%f\n", powf((var2 + bnEps), 0.5f));
	}

	//cnnBn更新
	dCnnBn[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)] = bnTmp * getDCnnBnGamma(outputIdx, cnnLayer) + getDCnnBnBeta(outputIdx, cnnLayer);
	/*
	if(outputIdx == 1){
		printf("cnnmean[%d]:%f\n", outputIdx , mean);
		printf("cnnvar2[%d]:%f\n", outputIdx , var2);
		printf("cnnbntmp(%d,%d,%d,%d,%d) = %f\n",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer, bnTmp);
		printf("cnnbn(%d,%d,%d,%d,%d) = %f\n",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer, dCnnBn[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)]);
	}
	*/
}

__global__ void kernelCnnBatchNormalizationOnline(int cnnLayer, float bnEps){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	float mean, var2, bnTmp;

	mean = getCCnnBnAveMean(outputIdx, cnnLayer);
	var2 = getCCnnBnAveVar2(outputIdx, cnnLayer);
	/*
	if(miniBatchIdx == 0 && wba_x == 0 && wba_y == 0){
		printf("cnnPropMean(%d)(%d):%f\n",outputIdx,cnnLayer,mean);
		printf("cnnPropVar2(%d)(%d):%f\n",outputIdx,cnnLayer, var2);
	}
	*/

	bnTmp = (getDCnnWb(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer) - mean) / powf((var2 + bnEps), 0.5f);

	//cnnBn更新
	dCnnBn[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)] = bnTmp * getDCnnBnGamma(outputIdx, cnnLayer) + getDCnnBnBeta(outputIdx, cnnLayer);
//	if(outputIdx == 1){
	//	printf("cnnmean[%d]:%f\n", outputIdx , mean);
	//	printf("cnnvar2[%d]:%f\n", outputIdx , var2);
	//	printf("cnnbntmp(%d,%d,%d,%d,%d) = %f  bnGamma() = %f  bnBeta() = %f bn = %f\n",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer, bnTmp, getDCnnBnGamma(outputIdx, cnnLayer), getDCnnBnBeta(outputIdx, cnnLayer), dCnnBn[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)]);
//	}
}
