#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>
#include <commonFunc.cuh>

__global__ void kernelBackCnnPoolingProp(const int cnnLayer);
extern __global__ void kernelBackCnnPoolingUpdate(const int cnnLayer, const int miniBatchIdxNums);

void backCnnPooling(const int cnnLayer){

//	puts("backCnnPooling start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);
	int inputChannelNums = getCnnOutputNums(cnnLayer + 1);

	//カーネルの次元設定
	dim3 gridProp(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 blockProp(getCnnP_xNums(cnnLayer), getCnnP_yNums(cnnLayer), 1); // 1ブロックcnnP_x * cnnP_yスレッド
	dim3 gridUpdate(getCnnOutputNums(cnnLayer), inputChannelNums); //cnnWの種類
	dim3 blockUpdate(getCnnW_xNums(), getCnnW_yNums(), 1); // 1ブロックcnnW_x * cnnW_yスレッド

	//次元チェック
	checkGridSize(gridProp);
	checkThreadSize(blockProp);
	checkGridSize(gridUpdate);
	checkThreadSize(blockUpdate);

	//シェアードメモリ確保
	int sharedSizeW = sizeof(float) * getCnnW_xNums() * getCnnW_yNums() * inputChannelNums; 

	//シェアードメモリチェック
	checkSharedMemorySize(sharedSizeW);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelBackCnnPoolingProp start.");
	hipDeviceSynchronize();
	kernelBackCnnPoolingProp<<<gridProp, blockProp, sharedSizeW>>>(cnnLayer);
//	puts("kernelBackCnnPoolingProp end.");
//	puts("kernelBackCnnPoolingUpdate start.");
	hipDeviceSynchronize();
	kernelBackCnnPoolingUpdate<<<gridUpdate, blockUpdate>>>(cnnLayer, getMiniBatchNums());
//	puts("kernelBackCnnPoolingUpdate end.");
//	gettimeofday(&t3, NULL);

//	puts("backCnnPooling end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelBackCnnPoolingProp(const int cnnLayer){
	int cnnP_x = threadIdx.x;
	int cnnP_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int cnnW_x, cnnW_y, input_x, input_y;
	float sum = 0;
	const int cnnLayerPlusOne = cnnLayer + 1;
	int inputIdx;
	int inputIdxNums = getCCnnOutputNums(cnnLayerPlusOne);
	int input_xNums = getCCnnWba_xNums(cnnLayerPlusOne);
	int input_yNums = getCCnnWba_yNums(cnnLayerPlusOne);

	//wをshared memoryにコピー
	extern __shared__ float sCnnW[];
	for(inputIdx = 0; inputIdx < inputIdxNums; inputIdx++){
		sCnnW[getDim3Idx(cnnP_x % getCCnnW_xNums(), cnnP_y % getCCnnW_yNums(), inputIdx, getCCnnW_xNums(), getCCnnW_yNums())]\
			= getDCnnW(cnnP_x % getCCnnW_xNums(), cnnP_y % getCCnnW_yNums(), outputIdx, inputIdx, cnnLayerPlusOne);
	}
	__syncthreads();

	//累積更新値取得:cnnP += cnnWb(0) * cnnW(0)
	for(inputIdx = 0; inputIdx < getCCnnOutputNums(cnnLayerPlusOne); inputIdx++){
		// 0 <= input_x,y < input_x,yNums の制約を持つ。
		//ただし、input_x,y < input_x,yNums は、for文中に書き込むとループ不足となってしまうため、別途制約クリアを実施
		for(cnnW_y = 0; cnnW_y < getCCnnW_yNums() && 0 <= cnnP_y - cnnW_y; cnnW_y++){
			for(cnnW_x = 0; cnnW_x < getCCnnW_xNums() && 0 <= cnnP_x - cnnW_x; cnnW_x++){
				input_x = cnnP_x - cnnW_x;
				input_y = cnnP_y - cnnW_y;
				sum += getDCnnWbBack(input_x, input_y, inputIdx, miniBatchIdx, cnnLayerPlusOne)\
				       * sCnnW[getDim3Idx(cnnW_x, cnnW_y, inputIdx, getCCnnW_xNums(), getCCnnW_yNums())]\
				       * (cnnP_y - cnnW_y < input_yNums) * (cnnP_x - cnnW_x < input_xNums); //後半の制約クリア
				/*
				if(outputIdx == 1 && cnnP_x == 1 &&cnnP_y == 2 && miniBatchIdx == 1){
					printf("sCnnW(%d,%d,%d,%d):%f\n", cnnW_x, cnnW_y, outputIdx, inputIdx, sCnnW[getDim3Idx(cnnW_x, cnnW_y, inputIdx, getCCnnW_xNums(), getCCnnW_yNums())]);
					printf("cnnWbBack(%d,%d,%d,%d):%f(sum=%f)\n", input_x, input_y, inputIdx, miniBatchIdx, getDCnnWbBack(input_x, input_y, inputIdx, miniBatchIdx, cnnLayerPlusOne),sum);
				}
				*/
			}
		}
	}
	//更新
	dCnnPBack[getDCnnPIdx(cnnP_x, cnnP_y, outputIdx, miniBatchIdx, cnnLayer)] = sum;
	/*
	if(outputIdx == 1 && cnnP_x == 1 &&cnnP_y == 2 && miniBatchIdx == 1){
		printf("cnnPBack(%d,%d,%d,%d)(%d):%f\n", cnnP_x, cnnP_y, outputIdx, miniBatchIdx, getDCnnPIdx(cnnP_x, cnnP_y, outputIdx, miniBatchIdx, cnnLayer),dCnnPBack[getDCnnPIdx(cnnP_x, cnnP_y, outputIdx, miniBatchIdx, cnnLayer)]);
	}
	*/
}

__global__ void
kernelBackCnnPoolingUpdate(const int cnnLayer, const int miniBatchIdxNums){
	int cnnW_x = threadIdx.x;
	int cnnW_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int inputIdx = blockIdx.y;
	int input_x, input_y, miniBatchIdx;
	float sum = 0.0;
	const int cnnLayerPlusOne = cnnLayer + 1;
	int input_xNums = getCCnnWba_xNums(cnnLayerPlusOne);
	int input_yNums = getCCnnWba_yNums(cnnLayerPlusOne);

	for(miniBatchIdx = 0; miniBatchIdx < miniBatchIdxNums; miniBatchIdx++){
		for(input_y = 0; input_y < input_yNums; input_y++){
			for(input_x = 0; input_x < input_xNums; input_x++){
				sum += getDCnnWbBack(input_x, input_y, inputIdx, miniBatchIdx, cnnLayerPlusOne)\
				       * getDCnnP(cnnW_x + input_x, cnnW_y + input_y, outputIdx, miniBatchIdx, cnnLayer);
				/*
				if(outputIdx == 1 && inputIdx == 1 &&cnnW_x == 1 && cnnW_y == 1){
					printf("cnnWbBack(%d,%d,%d,%d):%f\n", input_x, input_y, inputIdx, miniBatchIdx, getDCnnWbBack(input_x, input_y, inputIdx, miniBatchIdx, cnnLayerPlusOne));
					printf("cnnP(%d,%d,%d,%d):%f(%f)\n", cnnW_x + input_x, cnnW_y + input_y, outputIdx, miniBatchIdx, getDCnnP(cnnW_x + input_x, cnnW_y + input_y, outputIdx, miniBatchIdx, cnnLayer),sum);
				}
				*/
			}
		}
	}

	//更新
	dCnnW[getDCnnWIdx(cnnW_x, cnnW_y, outputIdx, inputIdx, cnnLayerPlusOne)] -= sum * getCLearningRate();
	//if(outputIdx == 1 && inputIdx == 1 &&cnnW_x == 1 && cnnW_y == 1){
//		printf("cnnW(%d,%d,%d,%d):%f  ", cnnW_x, cnnW_y, outputIdx, inputIdx, dCnnW[getDCnnWIdx(cnnW_x, cnnW_y, outputIdx, inputIdx, cnnLayerPlusOne)]);
	//}
}
