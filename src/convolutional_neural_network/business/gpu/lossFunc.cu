#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>


extern __global__ void kernelLossFunc(int mlpOutputNumsLastIdx);

void lossFunc(const int mlpOutputNumsLastIdx){

//	puts("lossFunc start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//ブロック・スレッド定義
	dim3 grid(getMlpOutputNums(mlpOutputNumsLastIdx), getMiniBatchNums());
	dim3 block(1, 1, 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelLossFunc start");
	hipDeviceSynchronize();
	kernelLossFunc<<<grid, block>>>(mlpOutputNumsLastIdx);

//	puts("kernelLossFunc end");
//	gettimeofday(&t3, NULL);

//	puts("lossFunc end.");
//	printTime(t1,t2,t3);

}

__global__ void kernelLossFunc(int mlpOutputNumsLastIdx){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;

	dMlpBnBack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpOutputNumsLastIdx)] = getDMlpA(outputIdx, miniBatchIdx, mlpOutputNumsLastIdx) - getDTeachOut(outputIdx, miniBatchIdx);
	/*
	if(miniBatchIdx == miniBatchIdx){
		printf("mlpBnBack(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpOutputNumsLastIdx, dMlpBnBack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpOutputNumsLastIdx)]);
	}
	*/
}
