#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

__global__ void kernelMlpBatchNormalizationTraining(int mlpLayer, float bnEps);
__global__ void kernelMlpBatchNormalizationOnline(int mlpLayer, float bnEps);

void mlpBatchNormalization(const int mlpLayer){

//	puts("mlpBatchNormalization start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//ブロック・スレッド定義
	dim3 grid(getMlpOutputNums(mlpLayer), getMiniBatchNums());
	dim3 block(1, 1, 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelMlpBatchNormalization start");
	hipDeviceSynchronize();
	if(getExecFlg() == getExecFlgTraining()){
		kernelMlpBatchNormalizationTraining<<<grid, block>>>(mlpLayer, getBnEps());
	}
	else{
		kernelMlpBatchNormalizationOnline<<<grid, block>>>(mlpLayer, getBnEps());
	}
//	puts("kernelMlpBatchNormalization end");
//	gettimeofday(&t3, NULL);

//	puts("mlpBatchNormalization end.");
//	printTime(t1,t2,t3);

}

__global__ void kernelMlpBatchNormalizationTraining(int mlpLayer, float bnEps){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int miniBatchIdxNums = gridDim.y;
	int z;
	float sumMean = 0, sumVar2 = 0;
	float tmp, mean, var2, bnTmp;

	//平均を算出
	for(z = 0; z < miniBatchIdxNums; z++){
		tmp = getDMlpWb(outputIdx, z, mlpLayer);
		/*
		if(outputIdx == 2 &&miniBatchIdx == 1){
			//printf("tmp:%f\n", tmp);
			printf("dMlpWb(%d,%d,%d)(%d) = %f\n", outputIdx, z, mlpLayer, getDMlpWbaIdx(outputIdx, z, mlpLayer),getDMlpWb(outputIdx, z, mlpLayer));
		}
		*/
		sumMean = tmp + sumMean;
	}
	mean = sumMean / (miniBatchIdxNums);

	//分散を算出
	for(z = 0; z < miniBatchIdxNums; z++){
		tmp = getDMlpWb(outputIdx, z, mlpLayer);
		sumVar2 += powf(floatSubtraction(tmp, mean, mlpLayer, __func__), 2);
	}
	var2 = sumVar2 / miniBatchIdxNums;
		
	/*
	if(miniBatchIdx == 0){
//		printf("mean:%f",mean);
		printf("var2:%f",var2);
	}
	*/
	dMlpBnMean[getDMlpBnMeanVar2Idx(outputIdx, mlpLayer)] = mean;
	dMlpBnVar2[getDMlpBnMeanVar2Idx(outputIdx, mlpLayer)] = var2;
	/*
	if(miniBatchIdx == 1){
//		printf("mlpPropMean[%d][%d]:%f\n", outputIdx , mlpLayer, mean);
//		printf("mlpPropVar2[%d][%d]:%f\n", outputIdx , mlpLayer, var2);
		printf("bnGamma(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, getDMlpBnGamma(outputIdx, mlpLayer));
		printf("bnBeta(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, getDMlpBnBeta(outputIdx, mlpLayer));
	}
	*/

	bnTmp = (getDMlpWb(outputIdx, miniBatchIdx, mlpLayer) - mean) / powf((var2 + bnEps), 0.5f);

	//mlpBn更新
	dMlpBn[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)] = bnTmp * getDMlpBnGamma(outputIdx, mlpLayer) + getDMlpBnBeta(outputIdx, mlpLayer);
	/*
	if(true){
		printf("bnTmp(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, bnTmp);
		printf("powf(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, powf((var2 + bnEps), 0.5f));
		printf("var2'(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, sumVar2 / (miniBatchIdxNums));
		printf("mean2'(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, (mean * mean));
		printf("bnGamma(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, getDMlpBnGamma(outputIdx, mlpLayer));
		printf("bnBeta(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, getDMlpBnBeta(outputIdx, mlpLayer));
	}
	if(miniBatchIdx == 0){
		printf("mlpBn(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, dMlpBn[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)]);
	}
	*/
}
__global__ void kernelMlpBatchNormalizationOnline(int mlpLayer, float bnEps){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	float mean, var2, bnTmp;

	mean = getCMlpBnAveMean(outputIdx, mlpLayer);
	var2 = getCMlpBnAveVar2(outputIdx, mlpLayer);
	/*
	if(miniBatchIdx == 0){
		printf("mlpmean(%d):%f\n",outputIdx,mean);
		printf("mlpvar2(%d):%f\n",outputIdx, var2);
	}
	*/

	bnTmp = (getDMlpWb(outputIdx, miniBatchIdx, mlpLayer) - mean) / powf((var2 + bnEps), 0.5f);
	if(powf((var2 + bnEps), 0.5f) <=0 ){
		printf("mlpkoko:%f\n", powf((var2 + bnEps), 0.5f));
	}

	//mlpBn更新
	dMlpBn[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)] = bnTmp * getDMlpBnGamma(outputIdx, mlpLayer) + getDMlpBnBeta(outputIdx, mlpLayer);
	/*
	if(miniBatchIdx == 0){
		printf("mlpBn(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, dMlpBn[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)]);
	}
	*/
}
