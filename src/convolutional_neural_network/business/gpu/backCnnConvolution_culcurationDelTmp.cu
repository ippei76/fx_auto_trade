#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

extern __global__ void kernelBackCnnCulcu1(const int cnnLayer, const float bnEps, const int targetFlg);
extern __global__ void kernelBackCnnCulcu2(const int cnnLayer, const int targetFlg);

void backCnnConvolution_culcurationDelTmp(const int cnnLayer, const int targetFlg){
	//targetFlg  0:delTmp 1:delGamma 2:delBeta

	//del2Tmp,del3Tmp
	//Culcu1###############################################################################################
	//ブロック・スレッド定義
	dim3 gridCulcu1(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 blockCulcu1(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1);

	//次元チェック
	checkGridSize(gridCulcu1);
	checkThreadSize(blockCulcu1);

	//シェアードメモリ確保
	int sharedSizeCnnWba_xyNums;
	sharedSizeCnnWba_xyNums = sizeof(float) * getCnnWba_xNums(cnnLayer) * getCnnWba_yNums(cnnLayer); 

	//シェアードメモリチェック
	checkSharedMemorySize(sharedSizeCnnWba_xyNums);

	hipDeviceSynchronize();
	//delTmpの計算
	kernelBackCnnCulcu1<<<gridCulcu1, blockCulcu1, sharedSizeCnnWba_xyNums>>>(cnnLayer, getBnEps(), targetFlg);
	//#####################################################################################################

	//Culcu2###############################################################################################
	//ブロック・スレッド定義
	dim3 gridCulcu2(getCnnOutputNums(cnnLayer), 1);
	dim3 blockCulcu2(getMiniBatchNums(), 1, 1);

	//次元チェック
	checkGridSize(gridCulcu2);
	checkThreadSize(blockCulcu2);

	//シェアードメモリ確保
	int sharedSizeMiniBatchNums;
	sharedSizeMiniBatchNums = sizeof(float) * getMiniBatchNums(); 

	//シェアードメモリチェック
	checkSharedMemorySize(sharedSizeMiniBatchNums);

	hipDeviceSynchronize();
	kernelBackCnnCulcu2<<<gridCulcu2, blockCulcu2, sharedSizeMiniBatchNums>>>(cnnLayer, targetFlg);
	//#####################################################################################################

}

//本来kernelでの分岐はふさわしくないが、ワープ内の全てのレーンで同じ処理を実施するため、速度低下は発生しないはず。
//http://news.mynavi.jp/series/kepler_gpu/002/

__global__ void kernelBackCnnCulcu1(const int cnnLayer, const float bnEps, const int targetFlg){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int miniBatchIdx = blockIdx.y;
	int outputIdx = blockIdx.x;
	int outputNums = gridDim.x;

	int threadIdxNo = getDim3Idx(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);
	int culcu1BlockIdxNo = getDim2Idx(outputIdx, miniBatchIdx, outputNums);

	float mean, var2;

	//平均値を取得
	mean = getDCnnBnMean(outputIdx, cnnLayer);
	//分散を取得
	var2 = getDCnnBnVar2(outputIdx, cnnLayer);

	//cnnWbをshared memoryにコピー
	extern __shared__ float sCnnBnBackWb[];

	//first
	if(targetFlg == 0){
		//del2Tmp
		sCnnBnBackWb[threadIdxNo] = getDCnnBnBack(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)\
				    * (getDCnnWb(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer) - mean) * powf((var2 + bnEps), -0.5f);
	}
	else if(targetFlg == 1){
		//delGamma
		float bnTmp = (getDCnnWb(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer) - mean) / powf((var2 + bnEps), 0.5f);
		sCnnBnBackWb[threadIdxNo] = getDCnnBnBack(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer) * bnTmp;
	}
	__threadfence_block();
	dCnnBackDelTmp1_culcu[culcu1BlockIdxNo] = culcurateSum(sCnnBnBackWb, threadIdxNo);

	__threadfence_block();
	//second
	if(targetFlg == 0){
		//del3Tmp
		sCnnBnBackWb[threadIdxNo] = getDCnnBnBack(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer);
	}
	else if(targetFlg == 1){
		//delBeta
		sCnnBnBackWb[threadIdxNo] = getDCnnBnBack(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer);
	}
	__threadfence_block();
	dCnnBackDelTmp2_culcu[culcu1BlockIdxNo] = culcurateSum(sCnnBnBackWb, threadIdxNo);
}

__global__ void kernelBackCnnCulcu2(const int cnnLayer, const int targetFlg){
	int miniBatchIdx = threadIdx.x;
	int outputIdx = blockIdx.x;
	int outputNums = gridDim.x;

	int threadIdxNo = getDim3Idx(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);
	int culcu1BlockIdxNo = getDim2Idx(outputIdx, miniBatchIdx, outputNums);

	extern __shared__ float sCnnDelTmp_culcu[];
	
	//delTmp1_culcuの結果を取得
	//common
	sCnnDelTmp_culcu[miniBatchIdx] = dCnnBackDelTmp1_culcu[culcu1BlockIdxNo];
	__threadfence_block();
	if(targetFlg == 0){
		//del2Tmp
		dCnnBackDel2Tmp[outputIdx] = culcurateSum(sCnnDelTmp_culcu, threadIdxNo);
	}
	else if(targetFlg == 1){
		//delGamma
		dCnnBackDelGamma[outputIdx] = culcurateSum(sCnnDelTmp_culcu, threadIdxNo);
	}
	__threadfence_block();

	//delTmp2_culcuの結果を取得
	//common
	sCnnDelTmp_culcu[miniBatchIdx] = dCnnBackDelTmp2_culcu[culcu1BlockIdxNo];
	__threadfence_block();
	if(targetFlg == 0){
		//del3Tmp
		dCnnBackDel3Tmp[outputIdx] = culcurateSum(sCnnDelTmp_culcu, threadIdxNo);
	}
	else if(targetFlg == 1){
		//delBeta
		dCnnBackDelBeta[outputIdx] = culcurateSum(sCnnDelTmp_culcu, threadIdxNo);
	}
}
