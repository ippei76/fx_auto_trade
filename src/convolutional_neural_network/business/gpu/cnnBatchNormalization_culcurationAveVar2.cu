#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

extern __global__ void kernelCnnCulcu1(const int cnnLayer, const int meanVar2Flg);
extern __global__ void kernelCnnCulcu2(const int cnnLayer, const int meanVar2Flg);

void cnnBatchNormalization_culcurationAveVar2(const int cnnLayer){

	//MEAN
	int meanFlg = 0;
	//Culcu1###############################################################################################
	//ブロック・スレッド定義
	dim3 gridCulcu1(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 blockCulcu1(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1);

	//次元チェック
	checkGridSize(gridCulcu1);
	checkThreadSize(blockCulcu1);

	//シェアードメモリ確保
	int sharedSizeCnnWba_xyNums;
	sharedSizeCnnWba_xyNums = sizeof(float) * getCnnWba_xNums(cnnLayer) * getCnnWba_yNums(cnnLayer); 

	//シェアードメモリチェック
	checkSharedMemorySize(sharedSizeCnnWba_xyNums);

	kernelCnnCulcu1<<<gridCulcu1, blockCulcu1, sharedSizeCnnWba_xyNums>>>(cnnLayer, meanFlg);
	hipDeviceSynchronize();
	//#####################################################################################################

	//Culcu2###############################################################################################
	//ブロック・スレッド定義
	dim3 gridCulcu2(getCnnOutputNums(cnnLayer), 1);
	dim3 blockCulcu2(getMiniBatchNums(), 1, 1);

	//次元チェック
	checkGridSize(gridCulcu2);
	checkThreadSize(blockCulcu2);

	//シェアードメモリ確保
	int sharedSizeMiniBatchNums;
	sharedSizeMiniBatchNums = sizeof(float) * getMiniBatchNums(); 

	//シェアードメモリチェック
	checkSharedMemorySize(sharedSizeMiniBatchNums);

	hipDeviceSynchronize();
	kernelCnnCulcu2<<<gridCulcu2, blockCulcu2, sharedSizeMiniBatchNums>>>(cnnLayer, meanFlg);
	//#####################################################################################################

	//VAR2
	int var2Flg = 1;
	//Culcu1###############################################################################################
	kernelCnnCulcu1<<<gridCulcu1, blockCulcu1, sharedSizeCnnWba_xyNums>>>(cnnLayer, var2Flg);
	hipDeviceSynchronize();
	//#####################################################################################################
	//Culcu2###############################################################################################
	hipDeviceSynchronize();
	kernelCnnCulcu2<<<gridCulcu2, blockCulcu2, sharedSizeMiniBatchNums>>>(cnnLayer, var2Flg);
	//#####################################################################################################
}

//本来kernelでの分岐はふさわしくないが、ワープ内の全てのレーンで同じ処理を実施するため、速度低下は発生しないはず。
//http://news.mynavi.jp/series/kepler_gpu/002/

__global__ void kernelCnnCulcu1(const int cnnLayer, const int meanVar2Flg){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int miniBatchIdx = blockIdx.y;
	int outputIdx = blockIdx.x;
	int outputNums = gridDim.x;

	int threadIdxNo = getDim3Idx(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);
	int threadNums = blockDim.x * blockDim.y * blockDim.z;
	int culcu1BlockIdxNo = getDim2Idx(outputIdx, miniBatchIdx, outputNums);

	/*
	if(blockIdx.x == 0 && blockIdx.y == 0){
		printf("dCnnWb(%d,%d,%d,%d,%d)(%d) = %f  ",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer, getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer),getDCnnWb(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer));
	}
	*/
	//cnnWbをshared memoryにコピー
	extern __shared__ float sCnnWb[];
	if(meanVar2Flg == 0){
		//MEAN
		sCnnWb[threadIdxNo] = getDCnnWb(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer);
	}
	else{
		//VAR2
		sCnnWb[threadIdxNo] = powf(floatSubtraction(getDCnnWb(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer), dCnnBnMean[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)], cnnLayer, __func__), 2);
	}
	__threadfence_block();
	dCnnBnAverage_culcu1[culcu1BlockIdxNo] = culcurateSum(sCnnWb, threadIdxNo) / threadNums;
}

__global__ void kernelCnnCulcu2(const int cnnLayer, const int meanVar2Flg){
	int miniBatchIdx = threadIdx.x;
	int outputIdx = blockIdx.x;
	int outputNums = gridDim.x;

	int threadIdxNo = getDim3Idx(threadIdx.x, threadIdx.y, threadIdx.z, blockDim.x, blockDim.y);
	int threadNums = blockDim.x * blockDim.y * blockDim.z;
	int culcu1BlockIdxNo = getDim2Idx(outputIdx, miniBatchIdx, outputNums);

	//cnnWbをshared memoryにコピー
	extern __shared__ float sCnnBnAverage_culcu1[];
	//culcu1の結果を取得
	sCnnBnAverage_culcu1[miniBatchIdx] = dCnnBnAverage_culcu1[culcu1BlockIdxNo];
	__threadfence_block();
	if(meanVar2Flg == 0){
		dCnnBnMean[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)] = culcurateSum(sCnnBnAverage_culcu1, threadIdxNo) / threadNums;
	}
	else{
		dCnnBnVar2[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)] = culcurateSum(sCnnBnAverage_culcu1, threadIdxNo) / threadNums;
	}

	/*
	if(blockIdx.x == 0 && blockIdx.y == 0){
		printf("culcu1 = %f  ", sCnnBnAverage_culcu1[miniBatchIdx]);
		printf("ave = %f  ", dCnnBnMean[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)]);
	}
	*/
}
