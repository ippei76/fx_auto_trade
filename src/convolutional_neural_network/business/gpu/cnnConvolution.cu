#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>
#include <sys/time.h>
#include <commonFunc.cuh>

extern __global__ void kernelCnnConvolutionSv();
extern __global__ void kernelCnnConvolutionCnnP(const int cnnLayer);

void cnnConvolution(const int cnnLayer){

//	puts("cnnConvolution start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//ブロック・スレッド定義
	//各miniBatchのwbノード毎にブロックを定義
	//wbノードの1要素毎にスレッドを定義
	//シェアードメモリにwを割り当てる
	dim3 grid(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 block(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

	//input,w相関チェック
	if(cnnLayer == 0){
		checkInputW(getSv_xNums(), getSv_yNums(), getCnnW_xNums(), getCnnW_yNums());
	}
	else{
		checkInputW(getCnnWba_xNums(cnnLayer - 1), getCnnWba_yNums(cnnLayer - 1), getCnnW_xNums(), getCnnW_yNums());
	}

	//シェアードメモリ確保
	int sharedSizeW;
	if(cnnLayer == 0){
		sharedSizeW = sizeof(float) * getCnnW_xNums() * getCnnW_yNums() * getSvChannelNums(); 
	}
	else{
		sharedSizeW = sizeof(float) * getCnnW_xNums() * getCnnW_yNums() * getCnnOutputNums(cnnLayer - 1); 
	}

	//シェアードメモリチェック
	checkSharedMemorySize(sharedSizeW);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelCnnConvolution start");
	if(cnnLayer == 0){
		kernelCnnConvolutionSv<<<grid, block, sharedSizeW>>>();
	}
	else{
		kernelCnnConvolutionCnnP<<<grid, block, sharedSizeW>>>(cnnLayer);
	}
//	puts("kernelCnnConvolution end");
//	gettimeofday(&t3, NULL);

//	puts("cnnConvolution end.");
	//printTime(t1,t2,t3);

}

__global__ void kernelCnnConvolutionSv(){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int x, y, inputIdx;
	float sum = 0;
	int inputChannelNums = getCSvChannelNums();
	int cnnLayerIsZero = 0;
	//wをshared memoryにコピー
	extern __shared__ float sCnnW[];
	for(inputIdx = 0; inputIdx < inputChannelNums; inputIdx++){
		sCnnW[getDim3Idx(wba_x % getCCnnW_xNums(), wba_y % getCCnnW_yNums(), inputIdx, getCCnnW_xNums(), getCCnnW_yNums())]\
			= getDCnnW(wba_x % getCCnnW_xNums(), wba_y % getCCnnW_yNums(), inputIdx, outputIdx, cnnLayerIsZero);
	}
	__syncthreads();

	//畳み込み処理
	for(inputIdx = 0; inputIdx < inputChannelNums; inputIdx++){
		for(y = 0; y < getCCnnW_yNums(); y++){
			for(x = 0; x < getCCnnW_xNums(); x++){
				//printf("x:%d\n",x);
				//printf("minib:%d,wba_x:%d,wba_y:%d,inputIdx:%d,outputIdx:%d\n",miniBatchIdx,wba_x,wba_y,inputIdx,outputIdx);
				sum += getDSv(x + wba_x, y + wba_y, inputIdx, miniBatchIdx)\
				       * sCnnW[getDim3Idx(x, y, inputIdx, getCCnnW_xNums(), getCCnnW_yNums())];
				
				/*
				if(miniBatchIdx == 0 && wba_x == 1 && wba_y == 1 && inputIdx == 0 && outputIdx == 1){
					//printf("x:%d\n",x);
					//printf("Sv(%d,%d,%d,%d,%d) = %f  ",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayerIsZero,getDSv(x + wba_x, y + wba_y, inputIdx, miniBatchIdx));
					//printf("cnnW(%d,%d,%d,%d,%d) = %f  ",x, y, inputIdx, outputIdx, cnnLayerIsZero,sCnnW[getDim3Idx(x, y, inputIdx, getCCnnW_xNums(), getCCnnW_yNums())]);
			//		printf("sum(%d,%d,%d) = %f\n",inputChannelNums,getCCnnW_yNums(),getCCnnW_xNums(),sum);
				}
				*/
				
			}
		}
	}
	__syncthreads();

	//wb更新
	dCnnWb[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayerIsZero)] = sum;
	//if(wba_x==6 && wba_y==6 &&outputIdx == 2 &&miniBatchIdx == 2){
	/*
	if(outputIdx == 0){
		printf("svwb(%d,%d,%d,%d,%d) = %f\n",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayerIsZero, sum);
	}
	*/
}

__global__ void kernelCnnConvolutionCnnP(const int cnnLayer){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int x, y, inputIdx;
	float sum = 0;
	int inputChannelNums = getCCnnOutputNums(cnnLayer - 1);
	//wをshared memoryにコピー
	extern __shared__ float sCnnW[];
	for(inputIdx = 0; inputIdx < inputChannelNums; inputIdx++){
		sCnnW[getDim3Idx(wba_x % getCCnnW_xNums(), wba_y % getCCnnW_yNums(), inputIdx, getCCnnW_xNums(), getCCnnW_yNums())]\
			= getDCnnW(wba_x % getCCnnW_xNums(), wba_y % getCCnnW_yNums(), inputIdx, outputIdx, cnnLayer);
	}
	__syncthreads();

	//畳み込み処理
	for(inputIdx = 0; inputIdx < inputChannelNums; inputIdx++){
		for(y = 0; y < getCCnnW_yNums(); y++){
			for(x = 0; x < getCCnnW_xNums(); x++){
				//printf("x:%d\n",x);
				sum += getDCnnP(x + wba_x, y + wba_y, inputIdx, miniBatchIdx, cnnLayer - 1)\
				       * sCnnW[getDim3Idx(x, y, inputIdx, getCCnnW_xNums(), getCCnnW_yNums())];
				//if(miniBatchIdx == 0 && wba_x == 1 && wba_y == 1 && inputIdx == 1 && outputIdx == 1){
				//if(miniBatchIdx == 0 && wba_x == 1 && wba_y == 1 && inputIdx == 1 && outputIdx == 1){
//					printf("p:%f  ",getDCnnP(x + wba_x, y + wba_y, inputIdx, miniBatchIdx, cnnLayer - 1));
				//	printf("scnnW:%f  ",sCnnW[getDim3Idx(x, y, inputIdx, getCCnnW_xNums(), getCCnnW_yNums())]);
				//}
			}
		}
	}
//	printf("cnnLayer:%d, wba_xNums:%d, getCCnnWba_xNums(cnnLayer):%d, getCCnnW_yNums():%d, inputChannelNums:%d\n", cnnLayer, blockDim.x,cWba_xNums[0], getCCnnW_yNums(), inputChannelNums);

	//wb更新
	dCnnWb[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)] = sum;
	//if(wba_x==17 && wba_y==16 &&outputIdx == 1 &&miniBatchIdx == 2){
	/*
	if(outputIdx == 1){
		printf("pwb(%d,%d,%d,%d,%d) = %f\n",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer, sum);
	}
	*/
}
