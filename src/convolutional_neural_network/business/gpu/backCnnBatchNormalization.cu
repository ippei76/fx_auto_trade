#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>
#include <commonFunc.cuh>

extern __device__ float reluDelA_bn(const int wba_x, const int wba_y, const int outputIdx, const int miniBatchIdx, const int cnnLayer);
extern __global__ void kernelBackCnnBatchNormalization(const int cnnLayer);

void backCnnBatchNormalization(const int cnnLayer){

//	puts("backCnnBatchNormalization start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//カーネルの次元設定
	dim3 grid(getCnnOutputNums(cnnLayer), getMiniBatchNums()); //miniBatch毎の1ニューロンの出力を1ブロックとする。
	dim3 block(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

//	gettimeofday(&t2, NULL);
//	puts("kernelBackCnnBatchNormalization start.");
	hipDeviceSynchronize();
	kernelBackCnnBatchNormalization<<<grid, block>>>(cnnLayer);
//	puts("kernelBackCnnBatchNormalization end.");
//	gettimeofday(&t3, NULL);

//	puts("backCnnBatchNormalization end.");

}

__global__ void
kernelBackCnnBatchNormalization(const int cnnLayer){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;

	dCnnBnBack[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)]\
		= getDCnnABack(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)\
		* reluDelA_bn(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer);
	/*
	if(miniBatchIdx == 1){
		printf("cnnABack(%d,%d,%d,%d,%d):%f\n", wba_x,wba_y,outputIdx, miniBatchIdx, cnnLayer, getDCnnABack(wba_x,wba_y,outputIdx, miniBatchIdx, cnnLayer));
		printf("cnnBn(%d,%d,%d,%d,%d):%f\n", wba_x,wba_y,outputIdx, miniBatchIdx, cnnLayer, getDCnnBn(wba_x,wba_y,outputIdx, miniBatchIdx, cnnLayer));
		printf("cnnBnBack(%d,%d,%d,%d,%d):%f\n", wba_x,wba_y,outputIdx, miniBatchIdx, cnnLayer, dCnnBnBack[getDCnnWbaIdx(wba_x,wba_y,outputIdx, miniBatchIdx, cnnLayer)]);
	}
	*/
}

__device__ float
reluDelA_bn(const int wba_x, const int wba_y, const int outputIdx, const int miniBatchIdx, const int cnnLayer){
	float val;
	val = (0 < getDCnnBn(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer));
	return(val);
}
