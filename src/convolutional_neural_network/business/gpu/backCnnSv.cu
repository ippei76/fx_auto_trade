#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>
#include <commonFunc.cuh>

extern __global__ void kernelBackCnnSvUpdate(const int miniBatchIdxNums);

void backCnnSv(const int cnnLayer){

//	puts("backCnnSv start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);
	int inputChannelNums = getCnnOutputNums(cnnLayer + 1);

	//カーネルの次元設定
	dim3 gridUpdate(getSvChannelNums(), inputChannelNums); //cnnWの種類
	dim3 blockUpdate(getCnnW_xNums(), getCnnW_yNums(), 1); // 1ブロックcnnW_x * cnnW_yスレッド

	//次元チェック
	checkGridSize(gridUpdate);
	checkThreadSize(blockUpdate);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelBackCnnSvUpdate start.");
	hipDeviceSynchronize();
	kernelBackCnnSvUpdate<<<gridUpdate, blockUpdate>>>(getMiniBatchNums());
	hipDeviceSynchronize();
//	puts("kernelBackCnnSvUpdate end.");
//	gettimeofday(&t3, NULL);

//	puts("backCnnSv end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelBackCnnSvUpdate(const int miniBatchIdxNums){
	int cnnW_x = threadIdx.x;
	int cnnW_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int inputIdx = blockIdx.y;
	int input_x, input_y, miniBatchIdx;
	float sum = 0.0;
	const int cnnLayerIsZero = 0;
	int input_xNums = getCCnnWba_xNums(cnnLayerIsZero);
	int input_yNums = getCCnnWba_yNums(cnnLayerIsZero);

	for(miniBatchIdx = 0; miniBatchIdx < miniBatchIdxNums; miniBatchIdx++){
		for(input_y = 0; input_y < input_yNums; input_y++){
			for(input_x = 0; input_x < input_xNums; input_x++){
				sum += getDCnnWbBack(input_x, input_y, inputIdx, miniBatchIdx, cnnLayerIsZero)\
				       * getDSv(cnnW_x + input_x, cnnW_y + input_y, outputIdx, miniBatchIdx);
				/*
				if(outputIdx == 0 && inputIdx == 0){
					printf("input_xNums():%d\n", input_xNums);
					printf("input_yNums():%d\n", input_yNums);
					printf("outputNums():%d\n", gridDim.x);
					printf("inputNums():%d\n", gridDim.y);
					printf("cnnWbBack(%d,%d,%d,%d):%f\n", input_x, input_y, inputIdx, miniBatchIdx, getDCnnWbBack(input_x, input_y, inputIdx, miniBatchIdx, cnnLayerIsZero));
					printf("cnnsv(%d,%d,%d,%d):%f(%f)\n", cnnW_x + input_x, cnnW_y + input_y, outputIdx, miniBatchIdx, getDSv(cnnW_x + input_x, cnnW_y + input_y, outputIdx, miniBatchIdx),sum);
				}
				*/
			}
		}
	}

	//更新
	dCnnW[getDCnnWIdx(cnnW_x, cnnW_y, outputIdx, inputIdx, cnnLayerIsZero)] -= sum * getCLearningRate();
	//printf("cnnUpdateSvW(%d,%d,%d,%d):%f  ", cnnW_x, cnnW_y, outputIdx, inputIdx, dCnnW[getDCnnWIdx(cnnW_x, cnnW_y, outputIdx, inputIdx, cnnLayerIsZero)]);
	//if(outputIdx == 0 && inputIdx == 1 &&cnnW_x == 1 && cnnW_y == 1){
	/*
	if(outputIdx == 0 && inputIdx == 0 &&cnnW_x == 0 && cnnW_y == 0){
		//printf("cnnW(%d,%d,%d,%d):%f  ", cnnW_x, cnnW_y, outputIdx, inputIdx, dCnnW[getDCnnWIdx(cnnW_x, cnnW_y, outputIdx, inputIdx, cnnLayerIsZero)]);
		printf("learningRate:%f  ", getCLearningRate());
	}
	*/
}
