#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

extern __global__ void kernelInferenceBnMeanVar2Cnn(float *d_infCnnBnMeanOutput, float *d_infCnnBnVar2Output);
extern __global__ void kernelInferenceBnMeanVar2Mlp(float *d_infMlpBnMeanOutput, float *d_infMlpBnVar2Output);

void inferenceBnMeanVar2(const int episode, float *infCnnBnMean, float *infCnnBnVar2, float *infMlpBnMean, float *infMlpBnVar2){

//	puts("inferenceBnMeanVar2 start.");

	float *d_infCnnBnMeanOutput;
	float *d_infCnnBnVar2Output;
	float *d_infMlpBnMeanOutput;
	float *d_infMlpBnVar2Output;

	//1エピソード分のサイズを計算
	int infCnnBnMeanVar2DataSize = sizeof(float) * getCnnOutputNumsSum();
	int infMlpBnMeanVar2DataSize = sizeof(float) * getMlpOutputNumsSum();
//	printf("epiNums:%d\n",episode);
//	printf("cnnsize:%d\n",infCnnBnMeanVar2DataSize);
//	printf("mlpsize:%d\n",infMlpBnMeanVar2DataSize);
	//GPUの動的確保
	gpuErrchk(hipMalloc((void**)&d_infCnnBnMeanOutput, infCnnBnMeanVar2DataSize));
	gpuErrchk(hipMalloc((void**)&d_infCnnBnVar2Output, infCnnBnMeanVar2DataSize));
	gpuErrchk(hipMalloc((void**)&d_infMlpBnMeanOutput, infMlpBnMeanVar2DataSize));
	gpuErrchk(hipMalloc((void**)&d_infMlpBnVar2Output, infMlpBnMeanVar2DataSize));

	//使用データをデバイスにコピー
	//何も必要なし。計算済みのmean,var2をホストに移すだけだからである。

	//cnnのブロック・スレッド定義
	dim3 gridCnn(getCnnOutputNumsSum(), 1, 1);
	dim3 blockCnn(1, 1, 1);
	//mlpのブロック・スレッド定義
	dim3 gridMlp(getMlpOutputNumsSum(), 1, 1);
	dim3 blockMlp(1, 1, 1);

	//次元チェック
	checkGridSize(gridCnn);
	checkThreadSize(blockCnn);
	checkGridSize(gridMlp);
	checkThreadSize(blockMlp);

	//カーネル起動
//	puts("kernelInferenceBnMeanVar2 start");
	kernelInferenceBnMeanVar2Cnn<<<gridCnn, blockCnn>>>(d_infCnnBnMeanOutput, d_infCnnBnVar2Output);
	kernelInferenceBnMeanVar2Mlp<<<gridMlp, blockMlp>>>(d_infMlpBnMeanOutput, d_infMlpBnVar2Output);
	hipDeviceSynchronize();
//	puts("kernelInferenceBnMeanVar2 end");

	//デバイスからホストへメモリ転送
	//GPUメモリより、更新されたデータをメモリにコピー
	float *h_infCnnBnMeanOutput; //host
	float *h_infCnnBnVar2Output; //host
	float *h_infMlpBnMeanOutput; //host
	float *h_infMlpBnVar2Output; //host
	//上記変数の動的確保
	h_infCnnBnMeanOutput = (float *)malloc(infCnnBnMeanVar2DataSize); 
	h_infCnnBnVar2Output = (float *)malloc(infCnnBnMeanVar2DataSize); 
	h_infMlpBnMeanOutput = (float *)malloc(infMlpBnMeanVar2DataSize);
	h_infMlpBnVar2Output = (float *)malloc(infMlpBnMeanVar2DataSize);
	//GPUからh_*にコピー
	gpuErrchk(hipMemcpy(h_infCnnBnMeanOutput, d_infCnnBnMeanOutput, infCnnBnMeanVar2DataSize, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_infCnnBnVar2Output, d_infCnnBnVar2Output, infCnnBnMeanVar2DataSize, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_infMlpBnMeanOutput, d_infMlpBnMeanOutput, infMlpBnMeanVar2DataSize, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_infMlpBnVar2Output, d_infMlpBnVar2Output, infMlpBnMeanVar2DataSize, hipMemcpyDeviceToHost));
	//inf*にh_*をコピー
	//infCnnBnMeanVar2DataSizeが常に一定であるため以下の通り記述できる。floatでわることを忘れずに。
	copyFloatArray(h_infCnnBnMeanOutput, infCnnBnMean, infCnnBnMeanVar2DataSize, (infCnnBnMeanVar2DataSize * episode / sizeof(float)));
	copyFloatArray(h_infCnnBnVar2Output, infCnnBnVar2, infCnnBnMeanVar2DataSize, (infCnnBnMeanVar2DataSize * episode / sizeof(float)));
	copyFloatArray(h_infMlpBnMeanOutput, infMlpBnMean, infMlpBnMeanVar2DataSize, (infMlpBnMeanVar2DataSize * episode / sizeof(float)));
	copyFloatArray(h_infMlpBnVar2Output, infMlpBnVar2, infMlpBnMeanVar2DataSize, (infMlpBnMeanVar2DataSize * episode / sizeof(float)));

	//メモリの解放
	free(h_infCnnBnMeanOutput);
	free(h_infCnnBnVar2Output);
	free(h_infMlpBnMeanOutput);
	free(h_infMlpBnVar2Output);
	gpuErrchk(hipFree(d_infCnnBnMeanOutput));
	gpuErrchk(hipFree(d_infCnnBnVar2Output));
	gpuErrchk(hipFree(d_infMlpBnMeanOutput));
	gpuErrchk(hipFree(d_infMlpBnVar2Output));

//	puts("inferenceBnMeanVar2 end.");

}

__global__ void kernelInferenceBnMeanVar2Cnn(float *d_infCnnBnMeanOutput, float *d_infCnnBnVar2Output){

	int cnnOutputNumsIdx = blockIdx.x;

	d_infCnnBnMeanOutput[cnnOutputNumsIdx] = dCnnBnMean[cnnOutputNumsIdx];
	d_infCnnBnVar2Output[cnnOutputNumsIdx] = dCnnBnVar2[cnnOutputNumsIdx];

}

__global__ void kernelInferenceBnMeanVar2Mlp(float *d_infMlpBnMeanOutput, float *d_infMlpBnVar2Output){

	int mlpOutputNumsIdx = blockIdx.x;

	d_infMlpBnMeanOutput[mlpOutputNumsIdx] = dMlpBnMean[mlpOutputNumsIdx];
	d_infMlpBnVar2Output[mlpOutputNumsIdx] = dMlpBnVar2[mlpOutputNumsIdx];

}
