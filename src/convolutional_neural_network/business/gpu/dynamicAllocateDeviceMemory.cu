#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>
#include <sys/time.h>
#include <commonFunc.cuh>


extern __global__ void kernelDynamicAllocateDeviceMemory(float *d_cnnW, float *d_mlpW);

void dynamicAllocateDeviceMemory(float *cnnW, const int cnnWDataNums, float *mlpW, const int mlpWDataNums){

	puts("dynamicAllocateDeviceMemory start.");
	struct timeval t1, t2, t3;
	gettimeofday(&t1, NULL);

	float *d_cnnW;
	float *d_mlpW;

	//サイズ算出
	int cnnWDataSize =  cnnWDataNums * sizeof(float)
	int mlpWDataSize =  mlpWDataNums * sizeof(float)

	//deviceメモリ確保
	gpuErrchk(hipMalloc((void**)&d_cnnW, cnnWDataSize));
	gpuErrchk(hipMalloc((void**)&d_mlpW, mlpWDataSize));

	//メモリコピー
	gpuErrchk(hipMemcpy(d_cnnW, cnnW, cnnWDataSize, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_mlpW, mlpW, mlpWDataSize, hipMemcpyHostToDevice));

	//ブロック・スレッド定義
	//各miniBatchのwbノード毎にブロックを定義
	//wbノードの1要素毎にスレッドを定義
	//シェアードメモリにwを割り当てる
	dim3 grid(1, 1);
	dim3 block(1, 1, 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

	gettimeofday(&t2, NULL);
	//カーネル起動
	puts("kernelDynamicAllocateDeviceMemory start");
	kernelDynamicAllocateDeviceMemory<<<grid, block>>>(d_cnnW, d_mlpW);
	hipDeviceSynchronize();
	puts("kernelDynamicAllocateDeviceMemory end");
	gettimeofday(&t3, NULL);

	//メモリの解放は最後に実施

	puts("dynamicAllocateDeviceMemory end.");
	printTime(t1,t2,t3);
}

__global__ void kernelDynamicAllocateDeviceMemory(float *d_cnnW, float *d_mlpW){
	dCnnW = d_cnnW;
	dMlpW = d_mlpW;
}
