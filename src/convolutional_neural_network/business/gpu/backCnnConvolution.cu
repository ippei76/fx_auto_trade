#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

extern __global__ void kernelBackCnnConvolutionProp(const int cnnLayer, const float bnEps);
extern __global__ void kernelBackCnnConvolutionUpdate(const int cnnLayer, const int wba_xNums, const int wba_yNums, const int miniBatchIdxNums, const float bnEps);
extern void backCnnConvolution_culcurationDelTmp(const int cnnLayer, const int targetFlg);

void backCnnConvolution(const int cnnLayer){

//	struct timeval t1, t2, t3;

	//ブロック・スレッド定義
	dim3 gridProp(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 blockProp(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1);
	dim3 gridUpdate(getCnnOutputNums(cnnLayer), 1);
	dim3 blockUpdate(1, 1, 1);

	//次元チェック
	checkGridSize(gridProp);
	checkThreadSize(blockProp);
	checkGridSize(gridUpdate);
	checkThreadSize(blockUpdate);

	//delTmpの事前計算
	backCnnConvolution_culcurationDelTmp(cnnLayer, 0);
	//カーネル起動
	hipDeviceSynchronize();
	kernelBackCnnConvolutionProp<<<gridProp, blockProp>>>(cnnLayer, getBnEps());

	//deltaBetaGammaの事前計算
//	backCnnConvolution_culcurationDelTmp(cnnLayer, 1);
	hipDeviceSynchronize();
	kernelBackCnnConvolutionUpdate<<<gridUpdate, blockUpdate>>>(cnnLayer, getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), getMiniBatchNums(), getBnEps());

//	gettimeofday(&t1, NULL);
//	gettimeofday(&t2, NULL);
//	gettimeofday(&t3, NULL);
//	printTime(t1,t2,t3);
//	exit(2);

}

__global__ void
kernelBackCnnConvolutionProp(const int cnnLayer, const float bnEps){
	int cnnWba_x = threadIdx.x;
	int cnnWba_y = threadIdx.y;
	int cnnWba_xNums = blockDim.x;
	int cnnWba_yNums = blockDim.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int miniBatchIdxNums = gridDim.y;
	float mean, var2;
	float del2Tmp = 0;
	float del3Tmp = 0;

	//平均値を取得
	mean = getDCnnBnMean(outputIdx, cnnLayer);
	//分散を取得
	var2 = getDCnnBnVar2(outputIdx, cnnLayer);
	/*
	if(miniBatchIdx == 0 && cnnWba_x == 0 && cnnWba_y == 0){
		printf("mean[%d,%d,%d]:%f\n", cnnWba_x,cnnWba_y,outputIdx , mean);
		printf("var2[%d,%d,%d]:%f\n", cnnWba_x,cnnWba_y,outputIdx , var2);
	}
	*/

	/*
	//cnnWb更新要素の計算
	int x, y, z;
	for(z = 0; z < miniBatchIdxNums; z++){
		for(y = 0; y < cnnWba_yNums; y++){
			for(x = 0; x < cnnWba_xNums; x++){
				del2Tmp += getDCnnBnBack(x, y, outputIdx, z, cnnLayer)\
					   * (getDCnnWb(x, y, outputIdx, z, cnnLayer) - mean) * powf((var2 + bnEps), -0.5f);
				//if(outputIdx == 1 && cnnWba_x == 1 &&cnnWba_y == 1 && miniBatchIdx == 0){
	//			if(x==17 && y==5 && z == 15 &&cnnWba_x == 17 &&cnnWba_y == 1 && miniBatchIdx == 1 && outputIdx == 1){
					//printf("cnnBnBack(%d,%d,%d,%d):%f\n", x, y, outputIdx, z, getDCnnBnBack(x,y,outputIdx, z, cnnLayer));
					//printf("cnnWb(%d,%d,%d,%d):%f\n", x,y,outputIdx, z, getDCnnWb(x,y,outputIdx, z, cnnLayer));
//					printf("sCnnTmp(%d,%d,%d,%d,%d):%f ", x,y,outputIdx, z, cnnLayer,getDCnnBnBack(x, y, outputIdx, z, cnnLayer) * (getDCnnWb(x, y, outputIdx, z, cnnLayer) - mean) * powf((var2 + bnEps), -0.5f));
	//			}
				del3Tmp += getDCnnBnBack(x, y, outputIdx, z, cnnLayer);
			}
		}
	}
	*/
	del2Tmp = dCnnBackDel2Tmp[outputIdx];
	del3Tmp = dCnnBackDel3Tmp[outputIdx];
	//mlpのminiBatchIdxNums → miniBatchIdxNums * cnnWba_x * cnnWba_y
	float del1 = (miniBatchIdxNums * cnnWba_xNums * cnnWba_yNums) * getDCnnBnBack(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer);
	float del2 = del2Tmp * (getDCnnWb(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer) - mean) * powf((var2 + bnEps), -0.5f);
	float del3 = del3Tmp;
	//float subtractDel = floatSubtraction(floatSubtraction(del1, del2), del3);
	float subtractDel = del1 - del2 - del3;

	/*
	if(cnnWba_x == 17 &&cnnWba_y == 5 && miniBatchIdx == 15 && outputIdx == 17){
		printf("del1(%d,%d,%d,%d,%d):%f\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, cnnLayer,del1);
		printf("del2(%d,%d,%d,%d,%d)%f\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, cnnLayer,del2);
		printf("del2Tmp(%d,%d,%d,%d,%d):%f(culcu:%f)\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, cnnLayer,del2Tmp, dCnnBackDel2Tmp[outputIdx]);
		printf("del3(%d,%d,%d,%d,%d):%f(culcu:%f)\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, cnnLayer,del3, dCnnBackDel3Tmp[outputIdx]);
	}
	*/

	dCnnWbBack[getDCnnWbaIdx(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer)] =\
		subtractDel * getDCnnBnGamma(outputIdx, cnnLayer) * powf((var2 + bnEps), -0.5f) / (miniBatchIdxNums * cnnWba_xNums * cnnWba_yNums);

//		printf("subtractDel(%d,%d,%d,%d):%f\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, subtractDel);
//		printf("cnnWbBack(%d,%d,%d,%d):%f\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, dCnnWbBack[getDCnnWbaIdx(cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, cnnLayer)]);
//		printf("del2Tmp(%d,%d,%d,%d,%d):%f(culcu:%f):%f\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, cnnLayer,del2Tmp, dCnnBackDel2Tmp[outputIdx],dCnnWbBack[getDCnnWbaIdx(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer)]);
//		printf("del3(%d,%d,%d,%d,%d):%f(culcu:%f):%f\n", cnnWba_x,cnnWba_y,outputIdx, miniBatchIdx, cnnLayer,del3, dCnnBackDel3Tmp[outputIdx],dCnnWbBack[getDCnnWbaIdx(cnnWba_x, cnnWba_y, outputIdx, miniBatchIdx, cnnLayer)]);
}

__global__ void
kernelBackCnnConvolutionUpdate(const int cnnLayer, const int wba_xNums, const int wba_yNums, const int miniBatchIdxNums, const float bnEps){

	int outputIdx = blockIdx.x;
	float mean, var2, bnTmp;
	float sumGamma = 0, sumBeta = 0;

	//平均値を取得
	mean = getDCnnBnMean(outputIdx, cnnLayer);
	//分散を取得
	var2 = getDCnnBnVar2(outputIdx, cnnLayer);

	//更新値を計算
	int x, y, z;
	for(z = 0; z < miniBatchIdxNums; z++){
		for(y = 0; y < wba_yNums; y++){
			for(x = 0; x < wba_xNums; x++){
				bnTmp = (getDCnnWb(x, y, outputIdx, z, cnnLayer) - mean) / powf((var2 + bnEps), 0.5f);
				sumGamma += getDCnnBnBack(x, y, outputIdx, z, cnnLayer) * bnTmp;
				sumBeta += getDCnnBnBack(x, y, outputIdx, z, cnnLayer);
				//if(outputIdx == 1){
//					printf("bnTmp(%d,%d,%d,%d):%f  ", x,y,outputIdx, z, bnTmp);
//					printf("sumGamma(%d,%d,%d,%d):%f  ", x,y,outputIdx, z, sumGamma);
//					printf("sumBeta(%d,%d,%d,%d):%f  ", x,y,outputIdx, z, sumBeta);
				//}

			}
		}
	}
	//sumGamma = dCnnBackDelGamma[outputIdx];
	//sumBeta = dCnnBackDelBeta[outputIdx];
	//更新
	dCnnBnGamma[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)] -= sumGamma * getCLearningRate();
	dCnnBnBeta[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)] -= sumBeta * getCLearningRate();
//	if(outputIdx == 1){
//		printf("cnnGamma(%d):%f  ", outputIdx, dCnnBnGamma[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)]);
//		printf("cnnBeta(%d):%f  ", outputIdx, dCnnBnBeta[getDCnnBnMeanVar2Idx(outputIdx, cnnLayer)]);
//	}
	/*
	printf("delGamma(%d,%d):%f(culcu:%f)\n", outputIdx, cnnLayer,sumGamma, dCnnBackDelGamma[outputIdx]);
	printf("delBeta(%d,%d):%f(culcu:%f):\n", outputIdx, cnnLayer, sumBeta, dCnnBackDelBeta[outputIdx]);
	*/
}
