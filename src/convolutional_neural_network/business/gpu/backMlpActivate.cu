#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>
#include <commonFunc.cuh>

__global__ void kernelBackMlpActivateProp(const int mlpLayer);
__global__ void kernelBackMlpActivateUpdate(const int mlpLayer);

void backMlpActivate(const int mlpLayer){

//	puts("backMlpActivate start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//カーネルの次元設定
	dim3 gridProp(getMlpOutputNums(mlpLayer), getMiniBatchNums()); //"mlpLayer"に注意
	dim3 blockProp(1, 1, 1); // 1ブロック1スレッド
	dim3 gridUpdate(getMlpOutputNums(mlpLayer), getMlpOutputNums(mlpLayer + 1)); //"mlpLayer"に注意
	dim3 blockUpdate(1, 1, 1); // 1ブロック1スレッド

	//次元チェック
	checkGridSize(gridProp);
	checkThreadSize(blockProp);
	checkGridSize(gridUpdate);
	checkThreadSize(blockUpdate);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelBackMlpActivate start.");
	hipDeviceSynchronize();
	kernelBackMlpActivateProp<<<gridProp, blockProp>>>(mlpLayer);
//	puts("kernelBackMlpActivate end.");
//	puts("kernelBackMlpActivateUpdate start.");
	hipDeviceSynchronize();
	kernelBackMlpActivateUpdate<<<gridUpdate, blockUpdate>>>(mlpLayer);
//	puts("kernelBackMlpActivateUpdate end.");
//	gettimeofday(&t3, NULL);

//	puts("backMlpActivate end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelBackMlpActivateProp(const int mlpLayer){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int inputIdx;
	float sum = 0;
	const int mlpLayerPlusOne = mlpLayer + 1;

	//累積更新値取得:mlpA += mlpWb(mlpLayer+1) * mlpW(mlpLayer+1)
	for(inputIdx = 0; inputIdx < getCMlpOutputNums(mlpLayerPlusOne); inputIdx++){
		//inputとoutputの割り当てがfowardpropのときと逆であることに注意
		sum += getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerPlusOne) * getDMlpW(outputIdx, inputIdx, mlpLayerPlusOne);
		/*
		if(miniBatchIdx == 2){
			printf("mlpWbBack(%d,%d,%d):%f\n", inputIdx, miniBatchIdx, mlpLayerPlusOne, getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerPlusOne));
			printf("mlpW(%d,%d,%d,%d):%f\n", outputIdx, inputIdx,miniBatchIdx, mlpLayerPlusOne, getDMlpW(outputIdx, inputIdx, mlpLayerPlusOne));
		}
		*/
	}
	//更新
	dMlpABack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)] = sum;
	/*
	if(miniBatchIdx == 2){
		printf("mlpABack(%d,%d,%d):%f\n", outputIdx, miniBatchIdx, mlpLayer, dMlpABack[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)]);
	}
	*/
}

__global__ void
kernelBackMlpActivateUpdate(const int mlpLayer){
	int outputIdx = blockIdx.x;
	int inputIdx = blockIdx.y;
	int miniBatchIdx;
	int miniBatchIdxNums = getCMiniBatchNums();
	float sum = 0;
	const int mlpLayerPlusOne = mlpLayer + 1;

	for(miniBatchIdx = 0; miniBatchIdx < miniBatchIdxNums; miniBatchIdx++){
		sum += getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerPlusOne) * getDMlpA(outputIdx, miniBatchIdx, mlpLayer);
		/*
		if(outputIdx == 2 && inputIdx == 1){
			printf("mlpWbBack(%d,%d,%d):%f\n", inputIdx, miniBatchIdx, mlpLayerPlusOne, getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerPlusOne));
			printf("mlpA(%d,%d,%d):%f\n", outputIdx, miniBatchIdx, mlpLayer, getDMlpA(outputIdx, miniBatchIdx, mlpLayer));
			printf("sum=%f\n",sum);
		}
		*/
	}

	//更新
	dMlpW[getDMlpWIdx(outputIdx, inputIdx, mlpLayerPlusOne)] -= sum * getCLearningRate();
	/*
	if(outputIdx == 2 && inputIdx == 1){
			printf("mlpW(%d,%d,%d,%d):%f\n", outputIdx, inputIdx,miniBatchIdx, mlpLayerPlusOne, getDMlpW(outputIdx, inputIdx, mlpLayerPlusOne));
			printf("mlpWidx:%d\n", getDMlpWIdx(outputIdx, inputIdx, mlpLayerPlusOne));
	}
	*/
}
