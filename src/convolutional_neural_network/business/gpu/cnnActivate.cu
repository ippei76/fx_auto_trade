#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>

extern __global__ void kernelCnnActivate(const int cnnLayer);
extern __device__ float relu(const int wba_x, const int wba_y, const int outputIdx, const int miniBatchIdx, const int cnnLayer);

void cnnActivate(const int cnnLayer){

//	puts("cnnActivate start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//カーネルの次元設定
	dim3 grid(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 block(getCnnWba_xNums(cnnLayer), getCnnWba_yNums(cnnLayer), 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

//	gettimeofday(&t2, NULL);
//	puts("kernelCnnActivate start.");
	hipDeviceSynchronize();
	//カーネル処理実行
	kernelCnnActivate<<<grid, block>>>(cnnLayer);
//	puts("kernelCnnActivate end.");
//	gettimeofday(&t3, NULL);

//	puts("cnnActivate end.");
//	printTime(t1,t2,t3);

}

__global__ void
kernelCnnActivate(const int cnnLayer){
	int wba_x = threadIdx.x;
	int wba_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;

	dCnnA[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)] = relu(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer);
	//if(wba_x==17 && wba_y==16 &&outputIdx == 1 &&miniBatchIdx == 2){
	//if(outputIdx == 1){
		//printf("cnnBn(%d,%d,%d,%d,%d) = %f  cnnA() = %f\n",wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer, getDCnnBn(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer), dCnnA[getDCnnWbaIdx(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer)]);
	//}
}

__device__ float
relu(const int wba_x, const int wba_y, const int outputIdx, const int miniBatchIdx, const int cnnLayer){
	float val;
	val = getDCnnBn(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer) *\
	      (0 < getDCnnBn(wba_x, wba_y, outputIdx, miniBatchIdx, cnnLayer));
	
	return(val);
}
