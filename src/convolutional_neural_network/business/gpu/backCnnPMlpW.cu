#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>
#include <commonFunc.cuh>

extern __global__ void kernelBackCnnPMlpWProp(const int cnnLastLayer);
extern __global__ void kernelBackCnnPMlpWUpdate(const int cnnLastLayer, const int miniBatchIdxNums);

void backCnnPMlpW(const int cnnLastLayer){

//	puts("backCnnPMlpW start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	int mlpLayerIsZero = 0;
	int inputChannelNums = mlpOutputNums[mlpLayerIsZero];
	//カーネルの次元設定
	dim3 gridProp(getCnnOutputNums(cnnLastLayer), getMiniBatchNums()); //"cnnLastLayer"に注意
	dim3 blockProp(getCnnP_xNums(cnnLastLayer), getCnnP_yNums(cnnLastLayer), 1); // 1ブロックp_x * p_yスレッド
	dim3 gridUpdate(getCnnOutputNums(cnnLastLayer), inputChannelNums); //"cnnLastLayer"に注意
	dim3 blockUpdate(getCnnP_xNums(cnnLastLayer), getCnnP_yNums(cnnLastLayer), 1); // 1ブロックp_x * p_yスレッド

	//次元チェック
	checkGridSize(gridProp);
	checkThreadSize(blockProp);
	checkGridSize(gridUpdate);
	checkThreadSize(blockUpdate);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelBackCnnPMlpWProp start.");
	hipDeviceSynchronize();
	kernelBackCnnPMlpWProp<<<gridProp, blockProp>>>(cnnLastLayer);
//	puts("kernelBackCnnPMlpWProp end.");
//	puts("kernelBackCnnPMlpWUpdate start.");
	hipDeviceSynchronize();
	kernelBackCnnPMlpWUpdate<<<gridUpdate, blockUpdate>>>(cnnLastLayer, getMiniBatchNums());
//	puts("kernelBackCnnPMlpWUpdate end.");
//	gettimeofday(&t3, NULL);

//	puts("backCnnPMlpW end.");
	//printTime(t1,t2,t3);

}

__global__ void
kernelBackCnnPMlpWProp(const int cnnLastLayer){
	int p_x = threadIdx.x;
	int p_xNums = blockDim.x;
	int p_y = threadIdx.y;
	int p_yNums = blockDim.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int inputIdx;
	float sum = 0;
	const int mlpLayerIsZero = 0;

	//累積更新値取得:cnnP += mlpWb(0) * mlpW(0)
	for(inputIdx = 0; inputIdx < getCMlpOutputNums(mlpLayerIsZero); inputIdx++){
		//inputとoutputの割り当てがfowardpropのときと逆であることに注意
		sum += getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerIsZero)\
		       * getDMlpW(getDim3Idx(p_x, p_y, outputIdx, p_xNums, p_yNums), inputIdx, mlpLayerIsZero);
		/*
		if(miniBatchIdx == 2){
			printf("mlpWbBack(%d,%d,%d):%f\n", inputIdx, miniBatchIdx, mlpLayerIsZero, getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerIsZero));
			printf("mlpW(%d,%d,%d,%d,%d,%d):%f\n", p_x,p_y,outputIdx, inputIdx,miniBatchIdx, mlpLayerIsZero, getDMlpW(getDim3Idx(p_x, p_y, outputIdx, p_xNums, p_yNums), inputIdx, mlpLayerIsZero));
		}
		*/
	}
	//更新
	dCnnPBack[getDCnnPIdx(p_x, p_y, outputIdx, miniBatchIdx, cnnLastLayer)] = sum;
	/*
	if(miniBatchIdx == 2){
		printf("cnnPBack(%d,%d,%d):%f\n", outputIdx, miniBatchIdx, cnnLastLayer, dCnnPBack[getDCnnPIdx(p_x, p_y, outputIdx, miniBatchIdx, cnnLastLayer)]);
	}
	*/
}

__global__ void
kernelBackCnnPMlpWUpdate(const int cnnLastLayer, const int miniBatchIdxNums){
	int p_x = threadIdx.x;
	int p_xNums = blockDim.x;
	int p_y = threadIdx.y;
	int p_yNums = blockDim.y;
	int outputIdx = blockIdx.x;
	int inputIdx = blockIdx.y;
	int miniBatchIdx;
	float sum = 0;
	const int mlpLayerIsZero = 0;

	for(miniBatchIdx = 0; miniBatchIdx < miniBatchIdxNums; miniBatchIdx++){
		sum += getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerIsZero)\
		       * getDCnnP(p_x, p_y, outputIdx, miniBatchIdx, cnnLastLayer);
		/*
		if(p_x==1 &&p_y==1 &&outputIdx == 1 && inputIdx == 1){
			printf("mlpWbBack(%d,%d,%d):%f\n", inputIdx, miniBatchIdx, mlpLayerIsZero, getDMlpWbBack(inputIdx, miniBatchIdx, mlpLayerIsZero));
			printf("P(%d,%d,%d,%d,%d):%f\n", p_x,p_y,outputIdx, miniBatchIdx, cnnLastLayer, getDCnnP(p_x, p_y, outputIdx, miniBatchIdx, cnnLastLayer));
			printf("sum=%f\n",sum);
		}
		*/
	}

	//更新
	dMlpW[getDMlpWIdx(getDim3Idx(p_x, p_y, outputIdx, p_xNums, p_yNums), inputIdx, mlpLayerIsZero)] -= sum * getCLearningRate();
	//if(p_x==1 &&p_y==1 &&outputIdx == 1 && inputIdx == 1){
//			printf("mlpW(%d,%d,%d,%d):%f  ", outputIdx, inputIdx,miniBatchIdx, mlpLayerIsZero, dMlpW[getDMlpWIdx(getDim3Idx(p_x, p_y, outputIdx, p_xNums, p_yNums), inputIdx, mlpLayerIsZero)]);
//	}
}
