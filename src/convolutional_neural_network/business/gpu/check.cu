#include "hip/hip_runtime.h"
#include <stdio.h>
#include <constraint.cuh>

extern void printResult();
extern void printVar2();
extern void printCnnMlpW();
extern void printCnnMlpBnBeta();
extern void printCnnMlpBnGamma();

void checkInput_xyNums(const int input_xNums, const int input_yNums){
	if(input_xNums > maxInput_xNums || input_yNums > maxInput_yNums){
		puts("LOG:input_xyNums error.");
		printf("LOG:input_xNums(%d) > maxInput_xNums(%d)\nor\n", input_xNums, maxInput_xNums);
		printf("LOG:input_yNums(%d) > maxInput_yNums(%d)", input_yNums, maxInput_yNums);
		exit(2);
	}
}

void checkInputChannelNums(const int inputChannelNums){
	if(inputChannelNums > maxInputChannelNums){
		puts("LOG:inputChannelNums error.");
		printf("LOG:inputChannelNums(%d) > maxInputChannelNums(%d)\nor\n", inputChannelNums, maxInputChannelNums);
		exit(2);
	}
}

void checkMiniBatchNums(const int miniBatchNums_arg){
	if(miniBatchNums_arg > maxMiniBatchNums){
		puts("LOG:miniBatchNums error.");
		printf("LOG:miniBatchNums(%d) > maxMiniBatchNums(%d)\nor\n", miniBatchNums_arg, maxMiniBatchNums);
		exit(2);
	}
}

void checkOutputNums(const int *cnnOutputNums_arg, const int cnnOutputNumsNums, const int *mlpOutputNums_arg, const int mlpOutputNumsNums){
	if(cnnOutputNumsNums > maxCnnOutputNumsNums || mlpOutputNumsNums > maxMlpOutputNumsNums){
		puts("LOG:outputNums error.");
		printf("LOG:cnnOutputNumsNums(%d) > maxCnnOutputNumsNums(%d)\nor\n", cnnOutputNumsNums, maxCnnOutputNumsNums);
		printf("LOG:mlpOutputNumsNums(%d) > maxMlpOutputNumsNums(%d)\nor\n", mlpOutputNumsNums, maxMlpOutputNumsNums);
		exit(2);
	}

	int layer;
	for(layer = 0; layer < cnnOutputNumsNums; layer++){
		if(cnnOutputNums_arg[layer] > maxCnnOutputNums){
			puts("LOG:outputNums error.");
			printf("LOG:cnnOutputNums[%d](%d) > maxCnnOutputNums(%d)\n", layer, cnnOutputNums_arg[layer], maxCnnOutputNums);
			exit(2);
		}
	}
	for(layer = 0; layer < mlpOutputNumsNums; layer++){
		if(mlpOutputNums_arg[layer] > maxMlpOutputNums){
			puts("LOG:outputNums error.");
			printf("LOG:mlpOutputNums[%d](%d) > maxMlpOutputNums(%d)\n", layer, mlpOutputNums_arg[layer], maxMlpOutputNums);
			exit(2);
		}
	}
}

void checkW_xyNums(const int w_xNums_arg, const int w_yNums_arg){
	if(w_xNums_arg > maxW_xNums || w_yNums_arg > maxW_yNums){
		puts("LOG:w_xyNums error.");
		printf("LOG:w_xNums(%d) > maxW_xNums(%d)\nor\n", w_xNums_arg, maxW_xNums);
		printf("LOG:w_yNums(%d) > maxW_yNums(%d)", w_yNums_arg, maxW_yNums);
		exit(2);
	}
}

void checkInputDataSize(const int inputDataSize){
	if(inputDataSize > maxInputDataNums * sizeof(float)){
		puts("LOG:inputDataSize error.");
		printf("LOG:inputDataSize : %d\n", inputDataSize);
		printf("LOG:maxInputDataNums : %d\n", maxInputDataNums);
		exit(2);
	}
}

void checkGridSize(dim3 dim3GridSize){
	int gridSize = dim3GridSize.x * dim3GridSize.y * dim3GridSize.z;
	if(gridSize > maxGridSize){ 
		puts("LOG:GridSize error.");
		printf("LOG:gridSize : %d\n", gridSize);
		printf("LOG:maxGridSize : %d\n", maxGridSize);
		exit(2);
	}
}

void checkThreadSize(dim3 dim3ThreadSize){
	int threadSize = dim3ThreadSize.x * dim3ThreadSize.y * dim3ThreadSize.z;
	if(threadSize > maxThreadSize){ 
		puts("LOG:ThreadSize error.");
		printf("LOG:threadSize : %d\n", threadSize);
		printf("LOG:maxThreadSize : %d\n", maxThreadSize);
		exit(2);
	}
	if(dim3ThreadSize.x > maxThreadSize){
		puts("LOG:dim3ThreadSize.x error.");
		printf("LOG:dim3ThreadSize.x : %d\n", dim3ThreadSize.x);
		printf("LOG:maxThreadSize_x : %d\n", maxThreadSize_x);
		exit(2);
	}
	if(dim3ThreadSize.y > maxThreadSize){
		puts("LOG:dim3ThreadSize.y error.");
		printf("LOG:dim3ThreadSize.y : %d\n", dim3ThreadSize.y);
		printf("LOG:maxThreadSize_y : %d\n", maxThreadSize_y);
		exit(2);
	}
	if(dim3ThreadSize.z > maxThreadSize){
		puts("LOG:dim3ThreadSize.z error.");
		printf("LOG:dim3ThreadSize.z : %d\n", dim3ThreadSize.z);
		printf("LOG:maxThreadSize_z : %d\n", maxThreadSize_z);
		exit(2);
	}
}

void checkSharedMemorySize(const int sharedMemorySize){
	if(sharedMemorySize > maxSharedMemorySize){ 
		puts("LOG:SharedMemorySize error.");
		printf("LOG:sharedMemorySize : %d\n", sharedMemorySize);
		printf("LOG:maxSharedMemorySize : %d\n", maxSharedMemorySize);
		exit(2);
	}
}

void checkInputW(const int input_xNums, const int input_yNums, const int w_xNums, const int w_yNums){
	if(input_xNums < w_xNums || input_yNums < w_yNums){
		puts("LOG:InputWSize error.");
		printf("LOG:input_xNums : %d\n", input_xNums);
		printf("LOG:w_xNums : %d\n", w_xNums);
		printf("LOG:input_yNums : %d\n", input_yNums);
		printf("LOG:w_yNums : %d\n", w_yNums);
		exit(2);
	}
	//畳み込み後のサイズがフィルタのサイズより小さくなってはいけない。
	//wをシェアードメモリにコピーするときに、コピー漏れが発生する。
	if(input_xNums - w_xNums + 1 < w_xNums || input_yNums - w_yNums + 1 < w_yNums){
		puts("LOG:Input and wSize error.");
		printf("LOG:input_xNums : %d\n", input_xNums);
		printf("LOG:w_xNums : %d\n", w_xNums);
		printf("LOG:input_yNums : %d\n", input_yNums);
		printf("LOG:w_yNums : %d\n", w_yNums);
		exit(2);
	}
}

void checkConstantMemory(){
	int constMemSum = 0;

	//cCnnOutputNums
	//cWba_xNums
	//cWba_yNums
	//cP_xNums
	//cP_yNums
	constMemSum += sizeof(int) * maxCnnOutputNumsNums * 5;

	//cMlpOutputNums
	constMemSum += sizeof(int) * maxMlpOutputNumsNums * 1;

	//cW_xNums
	//cW_yNums
	//cPooling_xNums
	//cPooling_yNums
	constMemSum += sizeof(int) * 4;

	//cCnnBnAveMean
	//cCnnBnAveVar2
	constMemSum += sizeof(int) * maxCnnLayerNumsSum * 2;

	//cMlpBnAveMean
	//cMlpBnAveVar2
	constMemSum += sizeof(int) * maxMlpLayerNumsSum * 2;

	if(constMemSum > maxConstantMemorySize){
		puts("LOG:constant memory size error.");
		printf("LOG:constMemSum(%d) > maxConstantMemorySize(%d)", constMemSum, maxConstantMemorySize);
		exit(2);
	}

}

void checkInfNan(const float inf_nan_arg, const char *message){
	if(isinf(inf_nan_arg)){
		printf("LOG:inf error.(%s)\n", message);
		printResult();
		printVar2();
		printCnnMlpW();
		printCnnMlpBnBeta();
		printCnnMlpBnGamma();
		exit(2);
	}
	if(isnan(inf_nan_arg)){
		printf("LOG:nan error.(%s)\n", message);
		printResult();
		printVar2();
		printCnnMlpW();
		printCnnMlpBnBeta();
		printCnnMlpBnGamma();
		exit(2);
	}
}

void checkNodeValues(const int cnnWbaDataNums, const int cnnPDataNums, const int mlpWbaDataNums, const int mlpLastOutputNums){
	if(cnnWbaDataNums > maxCnnWbaDataNums){
		puts("LOG:cnnWbaDataNums error.");
		printf("LOG:cnnWbaDataNums(%d) > max(%d)", cnnWbaDataNums, maxCnnWbaDataNums);
		exit(2);
	}
	if(cnnPDataNums > maxCnnPDataNums){
		puts("LOG:cnnPDataNums error.");
		printf("LOG:cnnPDataNums(%d) > max(%d)", cnnPDataNums, maxCnnPDataNums);
		exit(2);
	}
	if(mlpWbaDataNums > maxMlpWbaDataNums){
		puts("LOG:mlpWbaDataNums error.");
		printf("LOG:mlpWbaDataNums(%d) > max(%d)", mlpWbaDataNums, maxMlpWbaDataNums);
		exit(2);
	}
	if(mlpLastOutputNums > maxTeachOutNums){
		puts("LOG:mlpLastOutputNums error.");
		printf("LOG:mlpLastOutputNums(%d) > max(%d)", mlpLastOutputNums, maxTeachOutNums);
		exit(2);
	}
}

void checkLearnValues(const int cnnWDataNums, const int cnnBnBetaGammaDataNums, const int mlpWDataNums, const int mlpBnBetaGammaDataNums){
	if(cnnWDataNums > maxCnnWDataNums){
		puts("LOG:cnnWDataNums error.");
		printf("LOG:cnnWDataNums(%d) > max(%d)", cnnWDataNums, maxCnnWDataNums);
		exit(2);
	}
	if(cnnBnBetaGammaDataNums > maxCnnLayerNumsSum){
		puts("LOG:cnnBnBetaGammaDataNums error.");
		printf("LOG:cnnBnBetaGammaDataNums(%d) > max(%d)", cnnBnBetaGammaDataNums, maxCnnLayerNumsSum);
		exit(2);
	}
	if(mlpWDataNums > maxMlpWDataNums){
		puts("LOG:mlpWDataNums error.");
		printf("LOG:mlpWDataNums(%d) > max(%d)", mlpWDataNums, maxMlpWDataNums);
		exit(2);
	}
	if(mlpBnBetaGammaDataNums > maxMlpLayerNumsSum){
		puts("LOG:mlpBnBetaGammaDataNums error.");
		printf("LOG:mlpBnBetaGammaDataNums(%d) > max(%d)", mlpBnBetaGammaDataNums, maxMlpLayerNumsSum);
		exit(2);
	}

}

void checkSvTeachOutDataNums(const int oneSvDataNums, const int oneTeachOutDataNums){
	if(oneSvDataNums > maxInputDataNums){
		puts("LOG:oneSvDataNums error.");
		printf("LOG:oneSvDataNums(%d) > max(%d)", oneSvDataNums, maxInputDataNums);
		exit(2);
	}
	if(oneTeachOutDataNums > maxOutputDataNums){
		puts("LOG:oneTeachOutDataNums error.");
		printf("LOG:oneTeachOutDataNums(%d) > max(%d)", oneTeachOutDataNums, maxOutputDataNums);
		exit(2);
	}
}
