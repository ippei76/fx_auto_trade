#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh>

extern __global__ void kernelCnnPooling(const int cnnLayer);
extern __device__ void maxPooling(const int p_x, const int p_y, const int outputIdx, const int miniBatchIdx, const int cnnLayer);

void cnnPooling(const int cnnLayer){

//	puts("cnnPooling start.");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//カーネルの次元設定
	dim3 grid(getCnnOutputNums(cnnLayer), getMiniBatchNums());
	dim3 block(getCnnP_xNums(cnnLayer), getCnnP_yNums(cnnLayer), 1);

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

//	puts("kernelCnnPooling start.");
//	gettimeofday(&t2, NULL);
	//カーネル処理実行
	hipDeviceSynchronize();
	kernelCnnPooling<<<grid, block>>>(cnnLayer);
//	puts("kernelCnnPooling end.");
//	gettimeofday(&t3, NULL);

//	puts("cnnPooling end.");
//	printTime(t1,t2,t3);
}

__global__ void
kernelCnnPooling(const int cnnLayer){
	int p_x = threadIdx.x;
	int p_y = threadIdx.y;
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;

	maxPooling(p_x, p_y, outputIdx, miniBatchIdx, cnnLayer);
}

__device__ void
maxPooling(const int p_x, const int p_y, const int outputIdx, const int miniBatchIdx, const int cnnLayer){
	int x, y;
	float tmp;
	float max = getDCnnA(p_x * getCCnnPooling_xNums(), p_y * getCCnnPooling_yNums(), outputIdx, miniBatchIdx, cnnLayer);

	for(y = 0; y < getCCnnPooling_yNums() && y + p_y * getCCnnPooling_yNums() < getCCnnWba_yNums(cnnLayer); y++){
		for(x = 0; x < getCCnnPooling_xNums() && x + p_x * getCCnnPooling_xNums() < getCCnnWba_xNums(cnnLayer); x++){
				tmp = getDCnnA(x + p_x * getCCnnPooling_xNums(), y + p_y * getCCnnPooling_yNums(), outputIdx, miniBatchIdx, cnnLayer);
				max = max * (max >= tmp) + tmp * (max < tmp);
		}
	}

	dCnnP[getDCnnPIdx(p_x, p_y, outputIdx, miniBatchIdx, cnnLayer)] = max;
	//if(outputIdx == 1){
		//printf("p(%d,%d,%d,%d,%d) = %f\n",p_x, p_y, outputIdx, miniBatchIdx, cnnLayer, dCnnP[getDCnnPIdx(p_x, p_y, outputIdx, miniBatchIdx, cnnLayer)]);
//	}
}
