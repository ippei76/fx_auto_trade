#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <malloc.h>
#include <constraint.cuh>
#include <hostParameters.cuh>
#include <deviceParameters.cuh>
#include <check.cuh> 
#include <cudaCheck.cuh>

extern __global__ void kernelMultiplyCnnP();
extern __global__ void kernelMultiplyMlpA(const int mlpLayer);

void mlpMultiply(const int mlpLayer){

//	printf("multiply start.\n");
//	struct timeval t1, t2, t3;
//	gettimeofday(&t1, NULL);

	//ブロック・スレッド定義
	//各miniBatchのwbノード毎にブロックを定義
	//wbノードの1要素毎にスレッドを定義
	//シェアードメモリにwを割り当てる
	dim3 grid(getMlpOutputNums(mlpLayer), getMiniBatchNums()); //miniBatch毎の1ニューロンの出力を1ブロックとする。
	dim3 block(1, 1, 1); // 1ブロック1スレッド

	//次元チェック
	checkGridSize(grid);
	checkThreadSize(block);

//	gettimeofday(&t2, NULL);
	//カーネル起動
//	puts("kernelMultiply start");
	hipDeviceSynchronize();
	if(mlpLayer == 0){
		kernelMultiplyCnnP<<<grid, block>>>();
	}
	else{
		kernelMultiplyMlpA<<<grid, block>>>(mlpLayer);
	}
//	puts("kernelMultiply end");
//	gettimeofday(&t3, NULL);

//	puts("multiply end.");
//	printTime(t1,t2,t3);

}

__global__ void kernelMultiplyCnnP(){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int cnnLastLayerIdx = getCCnnOutputNumsNums() - 1;
	int x, y, inputIdx;
	int xNums = getCCnnP_xNums(cnnLastLayerIdx);
	int yNums = getCCnnP_yNums(cnnLastLayerIdx);
	int inputChannelNums = getCCnnOutputNums(cnnLastLayerIdx);
	int mlpLayerIsZero = 0;
	float sum = 0;

	//掛け合わせ処理
	for(inputIdx = 0; inputIdx < inputChannelNums; inputIdx++){
		for(y = 0; y < yNums; y++){
			for(x = 0; x < xNums; x++){
				sum += getDCnnP(x, y, inputIdx, miniBatchIdx, cnnLastLayerIdx)\
				       * getDMlpW(getDim3Idx(x, y, inputIdx, xNums, yNums), outputIdx, mlpLayerIsZero);
				/*
				if(miniBatchIdx == 1){
					printf("cnnP(%d,%d,%d):%f\n",x,y,inputIdx,getDCnnP(x, y, inputIdx, miniBatchIdx, cnnLastLayerIdx));
					printf("mlpW(%d,%d,%d):%f\n",x,y,inputIdx,getDMlpW(getDim3Idx(x, y, inputIdx, xNums, yNums), outputIdx, mlpLayerIsZero));
				}
				*/
			}
		}
	}

	//wb更新
	dMlpWb[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayerIsZero)] = sum;
	/*
	if(outputIdx == 1){
		printf("cnnpwb(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayerIsZero, sum);
	}
	*/
}
__global__ void kernelMultiplyMlpA(const int mlpLayer){
	int outputIdx = blockIdx.x;
	int miniBatchIdx = blockIdx.y;
	int inputIdx;
	int inputChannelNums = getCMlpOutputNums(mlpLayer - 1);
	float sum = 0;

	//掛け合わせ処理
	for(inputIdx = 0; inputIdx < inputChannelNums; inputIdx++){
		sum += getDMlpA(inputIdx, miniBatchIdx, mlpLayer - 1)\
		       * getDMlpW(inputIdx, outputIdx, mlpLayer);
		/*
		if(miniBatchIdx == 1){
			printf("mlpA(%d):%f\n",inputIdx,getDMlpA(inputIdx, miniBatchIdx, mlpLayer - 1));
			printf("mlpW(%d):%f\n",inputIdx,getDMlpW(inputIdx, outputIdx, mlpLayer));
		}
		*/
	}

	//wb更新
	dMlpWb[getDMlpWbaIdx(outputIdx, miniBatchIdx, mlpLayer)] = sum;
	/*
	if(outputIdx == 1){
		printf("cnnmlpAwb(%d,%d,%d) = %f\n",outputIdx, miniBatchIdx, mlpLayer, sum);
	}
	*/
}
